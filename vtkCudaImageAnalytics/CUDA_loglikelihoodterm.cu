#include "hip/hip_runtime.h"
#include "CUDA_loglikelihoodterm.h"
#include "CUDA_commonKernels.h"
#include "stdio.h"
#include "hip/hip_runtime.h"
#include "float.h"
#include "limits.h"

//#define DEBUG_VTKCUDA_ILLT

template void CUDA_ILLT_IncrementInformation<float>(float* labelData, float desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<double>(double* labelData, double desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<long>(long* labelData, long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<unsigned long>(unsigned long* labelData, unsigned long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<long long>(long long* labelData, long long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<unsigned long long>(unsigned long long* labelData, unsigned long long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<int>(int* labelData, int desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<unsigned int>(unsigned int* labelData, unsigned int desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<short>(short* labelData, short desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<unsigned short>(unsigned short* labelData, unsigned short desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<char>(char* labelData, char desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<signed char>(signed char* labelData, signed char desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<unsigned char>(unsigned char* labelData, unsigned char desiredValue, short* agreement, int size, hipStream_t* stream);

template< class T >
void CUDA_ILLT_IncrementInformation(T* labelData, T desiredValue, short* agreement, int size, hipStream_t* stream){
    T* GPUBuffer = 0;

	hipMalloc((void**) &GPUBuffer, sizeof(T)*size);
	hipMemcpyAsync( GPUBuffer, labelData, sizeof(T)*size, hipMemcpyHostToDevice, *stream );

	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_IncrementInformation: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif

	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	IncrementBuffer<T><<<grid,threads,0,*stream>>>(GPUBuffer, desiredValue, agreement, size);
	hipFree(GPUBuffer);

	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_IncrementInformation: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

void CUDA_ILLT_GetRelevantBuffers(short** agreement, int size, hipStream_t* stream){
	hipMalloc((void**) agreement, sizeof(short)*size);
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	ZeroOutBuffer<short><<<grid,threads,0,*stream>>>(*agreement,size);

	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_GetRelevantBuffers: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

void CUDA_ILLT_CopyBackResult(float* GPUBuffer, float* CPUBuffer, int size, hipStream_t* stream){
	hipDeviceSynchronize();
	hipMemcpy( CPUBuffer, GPUBuffer, sizeof(float)*size, hipMemcpyDeviceToHost );
	hipFree(GPUBuffer);

	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_CopyBackResult: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

void CUDA_ILLT_AllocateHistogram(float** histogramGPU, int size, hipStream_t* stream){
	hipMalloc((void**) histogramGPU, sizeof(float)*size);
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_AllocateHistogram: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

void CUDA_ILLT_ReturnBuffer(float* buffer){
	hipFree(buffer);
}

template void CUDA_ILLT_CalculateHistogramAndTerms<double>(float* outputBuffer, float* histogramGPU, short* agreement, double* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<long>(float* outputBuffer, float* histogramGPU, short* agreement, long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<unsigned long>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<long long>(float* outputBuffer, float* histogramGPU, short* agreement, long long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<unsigned long long>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned long long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<int>(float* outputBuffer, float* histogramGPU, short* agreement, int* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<unsigned int>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned int* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<short>(float* outputBuffer, float* histogramGPU, short* agreement, short* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<unsigned short>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned short* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<char>(float* outputBuffer, float* histogramGPU, short* agreement, char* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<signed char>(float* outputBuffer, float* histogramGPU, short* agreement, signed char* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<unsigned char>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned char* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<float>(float* outputBuffer, float* histogramGPU, short* agreement, float* image,	 short requiredAgreement, int imageSize, hipStream_t* stream);


template void CUDA_ILLT_CalculateHistogramAndTerms2D<double>(float* outputBuffer, float* histogramGPU, short* agreement, double* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<long>(float* outputBuffer, float* histogramGPU, short* agreement, long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<unsigned long>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<long long>(float* outputBuffer, float* histogramGPU, short* agreement, long long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<unsigned long long>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned long long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<int>(float* outputBuffer, float* histogramGPU, short* agreement, int* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<unsigned int>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned int* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<short>(float* outputBuffer, float* histogramGPU, short* agreement, short* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<unsigned short>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned short* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<char>(float* outputBuffer, float* histogramGPU, short* agreement, char* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<signed char>(float* outputBuffer, float* histogramGPU, short* agreement, signed char* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<unsigned char>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned char* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms2D<float>(float* outputBuffer, float* histogramGPU, short* agreement, float* image,	 short requiredAgreement, int imageSize, hipStream_t* stream);

template<class T>
__global__ void kern_PopulateWorkingUp(float* working, short* agreement, T* image, short requiredAgreement, int imageSize){
	int idx = CUDASTDOFFSET;
	float inputValue = (float) image[idx];
	short lAgreement = agreement[idx];
	float outputValue = (lAgreement < requiredAgreement) ? FLT_MIN: inputValue;
	if(idx < imageSize) working[idx] = outputValue;
}

template<class T>
__global__ void kern_PopulateWorkingDown(float* working, short* agreement, T* image, short requiredAgreement, int imageSize){
	int idx = CUDASTDOFFSET;
	float inputValue = (float) image[idx];
	short lAgreement = agreement[idx];
	float outputValue = (lAgreement < requiredAgreement) ? FLT_MAX: inputValue;
	if(idx < imageSize) working[idx] = outputValue;
}

__global__ void kern_PropogateUp(float* working, int span, int imageSize){
	int idx = CUDASTDOFFSET;
	float inputValue1 = working[idx];
	float inputValue2 = working[idx+span];
	float outputVal = (inputValue1 > inputValue2) ? inputValue1: inputValue2;
	if(idx+span < imageSize) working[idx] = outputVal;
}

__global__ void kern_PropogateDown(float* working, int span, int imageSize){
	int idx = CUDASTDOFFSET;
	float inputValue1 = working[idx];
	float inputValue2 = working[idx+span];
	float outputVal = (inputValue1 < inputValue2) ? inputValue1: inputValue2;
	if(idx+span < imageSize) working[idx] = outputVal;
}

template<class T>
__global__ void kern_PopulateHisto(float* histogramGPU, short* agreement, T* image, short requiredAgreement, float imMin, float imMax, int imageSize){
	__shared__ float histogram[NUMTHREADS];
	int idx = threadIdx.x;
	imMin -= (imMax-imMin)*0.00625;
	imMax += (imMax-imMin)*0.00625;

	histogram[idx] = 1e-10f;
	__syncthreads();
	int repetitions = (imageSize-1) / blockDim.x + 1;
	int idxCurr = idx;
	for(int i = 0; i < repetitions; i++, idxCurr += blockDim.x){
		short localAgreement = agreement[idxCurr];
		float localValue = (float) image[idxCurr];
		int histInPos = (int) ( (float) (NUMTHREADS-1) * ((localValue-imMin) / (imMax-imMin)) + 0.5f );
		int histPos = idx;
		for(int h = 0; h < NUMTHREADS; h++){
			__syncthreads();
			histogram[histPos] += (idxCurr < imageSize && localAgreement >= requiredAgreement && histPos == histInPos) ? 1 : 0;
			histPos += (histPos < NUMTHREADS-1) ? 1: -histPos;	
		}
	}
	__syncthreads();

	//normalize inefficiently
	if(idx==0){
		float sum = 0.0f;
		for(int h = 0; h < NUMTHREADS; h++)
			sum += histogram[h];
		for(int h = 0; h < NUMTHREADS; h++)
			histogram[h] /= sum;
	}
	
	__syncthreads();
	histogramGPU[idx] = histogram[idx];

}

template<class T>
__global__ void kern_PopulateHisto2D(float* histogramGPU, short* agreement, T* image, short requiredAgreement, float imMin, float imMax, float sMin, float sMax, int imageSize){
	__shared__ float histogram[NUMTHREADS];
	int idx = threadIdx.x;
	imMin -= (imMax-imMin)*0.00625;
	imMax += (imMax-imMin)*0.00625;

	histogram[idx] = 1e-10f;
	__syncthreads();
	int repetitions = (imageSize-1) / blockDim.x + 1;
	int idxCurr = idx;
	for(int i = 0; i < repetitions; i++, idxCurr += blockDim.x){
		short localAgreement = agreement[idxCurr];
		float localValue1 = (float) image[2*idxCurr];
		float localValue2 = (float) image[2*idxCurr+1];
		int histInPos = (int) ( (float) (NUMTHREADS-1) * ((localValue1-imMin) / (imMax-imMin)) + 0.5f );
		int histPos = idx;
		bool useIt = (idxCurr < imageSize && localAgreement >= requiredAgreement && localValue2 >= sMin && localValue2 < sMax);
		for(int h = 0; h < NUMTHREADS; h++){
			__syncthreads();
			histogram[histPos] += (useIt && histPos == histInPos) ? 1 : 0;
			histPos += (histPos < NUMTHREADS-1) ? 1: -histPos;	
		}
	}
	__syncthreads();
	histogramGPU[idx] = histogram[idx];

}

template<class T>
__global__ void kern_PopulateOutput(float* histogramGPU, float* output, T* image, float imMin, float imMax, int imageSize){
	__shared__ float histogram[NUMTHREADS];
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	imMin -= (imMax-imMin)*0.00625;
	imMax += (imMax-imMin)*0.00625;
	if( threadIdx.x < NUMTHREADS ) histogram[threadIdx.x] = histogramGPU[threadIdx.x];
	__syncthreads();
	
	float localValue = (float) image[idx];
	int histPos = (int) ( (float) (NUMTHREADS-1) * ((localValue-imMin) / (imMax-imMin)) + 0.5f );
	float histVal = (histPos < NUMTHREADS && histPos >= 0) ? histogram[histPos] : 1e-10f;
	histVal = (histVal < 1e-10f) ? 1e-10f : histVal;
	histVal = log(histVal) / log(1e-10f);
	if(idx < imageSize) output[idx] = histVal;

}

template<class T>
__global__ void kern_PopulateOutput2D(float* histogramGPU, float* output, T* image, float imMin, float imMax, float sMin, float sMax, int imageSize){
	__shared__ float histogram[NUMTHREADS];
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	imMin -= (imMax-imMin)*0.00625;
	imMax += (imMax-imMin)*0.00625;
	if( threadIdx.x < NUMTHREADS ) histogram[threadIdx.x] = histogramGPU[threadIdx.x];
	__syncthreads();
	
	float localValue1 = (float) image[2*idx];
	float localValue2 = (float) image[2*idx+1];
	int histPos = (int) ( (float) (NUMTHREADS-1) * ((localValue1-imMin) / (imMax-imMin)) + 0.5f );
	bool useIt = (localValue2 >= sMin && localValue2 < sMax);
	float histVal = (histPos < NUMTHREADS && histPos >= 0) ? histogram[histPos] : 1e-10f;
	histVal = (histVal < 1e-10f) ? 1e-10f : histVal;
	histVal = log(histVal) / log(1e-10f);
	float oldHistVal = output[idx];
	histVal = useIt ? histVal: oldHistVal;
	if(idx < imageSize) output[idx] = histVal;

}


template< class T >
void CUDA_ILLT_CalculateHistogramAndTerms(float* outputBuffer, float* histogramGPU, short* agreement, T* image, short requiredAgreement, int imageSize, hipStream_t* stream){
	
	T* GPUInputBuffer = 0;
	float* GPUOutputBuffer = 0;
	float* GPUWorkingBuffer = 0;
	hipMalloc((void**) &GPUInputBuffer, sizeof(T)*imageSize);
	hipMalloc((void**) &GPUOutputBuffer, sizeof(float)*imageSize);
	hipMalloc((void**) &GPUWorkingBuffer, sizeof(float)*imageSize);
	hipMemcpyAsync( GPUInputBuffer, image, sizeof(T)*imageSize, hipMemcpyHostToDevice, *stream );

	float imMax = 0;
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid = GetGrid(imageSize);
	kern_PopulateWorkingUp<T><<<grid,threads,0,*stream>>>(GPUWorkingBuffer, agreement, GPUInputBuffer, requiredAgreement, imageSize);
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_CalculateMinMax: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
	for(int t = (imageSize-1)/2+1; t > 0; t/=2){
		threads = dim3(NUMTHREADS,1,1);
		grid = GetGrid(t);
		kern_PropogateUp<<<grid,threads,0,*stream>>>(GPUWorkingBuffer, t, imageSize);

		#ifdef DEBUG_VTKCUDA_ILLT
			hipDeviceSynchronize();
			printf( "CUDA_ILLT_CalculateMinMax: " );
			printf( hipGetErrorString( hipGetLastError() ) );
			printf( "\n" );
		#endif
	}
	hipMemcpyAsync( &imMax, GPUWorkingBuffer, sizeof(float), hipMemcpyDeviceToHost, *stream );
	hipDeviceSynchronize();

	float imMin = 0;
	threads = dim3(NUMTHREADS,1,1);
	grid = dim3( (imageSize-1)/NUMTHREADS+1, 1, 1);
	kern_PopulateWorkingDown<T><<<grid,threads,0,*stream>>>(GPUWorkingBuffer, agreement, GPUInputBuffer, requiredAgreement, imageSize);
	for(int t = (imageSize-1)/2+1; t > 0; t/=2){
		threads = dim3(NUMTHREADS,1,1);
		grid = GetGrid(t);
		kern_PropogateDown<<<grid,threads,0,*stream>>>(GPUWorkingBuffer, t, imageSize);

		
		#ifdef DEBUG_VTKCUDA_ILLT
			hipDeviceSynchronize();
			printf( "CUDA_ILLT_CalculateMinMax: " );
			printf( hipGetErrorString( hipGetLastError() ) );
			printf( "\n" );
		#endif
	}
	hipMemcpyAsync( &imMin, GPUWorkingBuffer, sizeof(float), hipMemcpyDeviceToHost, *stream );
	hipDeviceSynchronize();
	

	threads = dim3(NUMTHREADS,1,1);
	grid = dim3( 1, 1, 1);
	kern_PopulateHisto<T><<<grid,threads,0,*stream>>>(histogramGPU, agreement, GPUInputBuffer, requiredAgreement, imMax, imMin, imageSize);
	
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_CalculateHistogram: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif

	grid = GetGrid(imageSize);
	kern_PopulateOutput<T><<<grid,threads,0,*stream>>>(histogramGPU, GPUOutputBuffer, GPUInputBuffer, imMax, imMin, imageSize);

	hipMemcpyAsync( outputBuffer, GPUOutputBuffer, sizeof(float)*imageSize, hipMemcpyDeviceToHost, *stream );
	
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_CalculateTerms: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif

	hipFree(GPUOutputBuffer);
	hipFree(GPUInputBuffer);
	hipFree(GPUWorkingBuffer);
	hipFree(histogramGPU);
	hipFree(agreement);
	
}

template< class T >
void CUDA_ILLT_CalculateHistogramAndTerms2D(float* outputBuffer, float* histogramGPU, short* agreement, T* image, short requiredAgreement, int imageSize, hipStream_t* stream){
	
	T* GPUInputBuffer = 0;
	float* GPUOutputBuffer = 0;
	float* GPUWorkingBuffer = 0;
	hipMalloc((void**) &GPUInputBuffer, 2*sizeof(T)*imageSize);
	hipMalloc((void**) &GPUOutputBuffer, sizeof(float)*imageSize);
	hipMalloc((void**) &GPUWorkingBuffer, sizeof(float)*imageSize);
	hipMemcpyAsync( GPUInputBuffer, image, 2*sizeof(T)*imageSize, hipMemcpyHostToDevice, *stream );

	float2 imMax = {0.0f, 0.0f};
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid = GetGrid(imageSize);
	kern_PopulateWorkingUp<T><<<grid,threads,0,*stream>>>(GPUWorkingBuffer, agreement, GPUInputBuffer, requiredAgreement, imageSize);
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_CalculateMinMax: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
	for(int t = (imageSize-1)/2+1; t > 1; t/=2){
		threads = dim3(NUMTHREADS,1,1);
		grid = GetGrid(t);
		kern_PropogateUp<<<grid,threads,0,*stream>>>(GPUWorkingBuffer, t, imageSize);

		#ifdef DEBUG_VTKCUDA_ILLT
			hipDeviceSynchronize();
			printf( "CUDA_ILLT_CalculateMinMax: " );
			printf( hipGetErrorString( hipGetLastError() ) );
			printf( "\n" );
		#endif
	}
	hipMemcpyAsync( &imMax, GPUWorkingBuffer, sizeof(float2), hipMemcpyDeviceToHost, *stream );
	hipDeviceSynchronize();

	float2 imMin = {0.0f, 0.0f};
	threads = dim3(NUMTHREADS,1,1);
	grid = dim3( (imageSize-1)/NUMTHREADS+1, 1, 1);
	kern_PopulateWorkingDown<T><<<grid,threads,0,*stream>>>(GPUWorkingBuffer, agreement, GPUInputBuffer, requiredAgreement, imageSize);
	for(int t = (imageSize-1)/2+1; t > 1; t/=2){
		threads = dim3(NUMTHREADS,1,1);
		grid = GetGrid(t);
		kern_PropogateDown<<<grid,threads,0,*stream>>>(GPUWorkingBuffer, t, imageSize);

		#ifdef DEBUG_VTKCUDA_ILLT
			hipDeviceSynchronize();
			printf( "CUDA_ILLT_CalculateMinMax: " );
			printf( hipGetErrorString( hipGetLastError() ) );
			printf( "\n" );
		#endif
	}
	hipMemcpyAsync( &imMin, GPUWorkingBuffer, sizeof(float2), hipMemcpyDeviceToHost, *stream );
	hipDeviceSynchronize();
	
	//populate unnormalized histogram
	threads = dim3(NUMTHREADS,1,1);
	grid = dim3( 1, 1, 1);
	for(int comp = 0; comp < NUMTHREADS; comp++){
		float secondMin = imMin.y + (float) comp * (imMax.y-imMin.y) / (float) NUMTHREADS;
		float secondMax = (comp != NUMTHREADS-1) ? imMin.y + (float) (comp+1) * (imMax.y-imMin.y) / (float) NUMTHREADS : FLT_MAX;
		kern_PopulateHisto2D<T><<<grid,threads,0,*stream>>>(histogramGPU+comp*NUMTHREADS, agreement, GPUInputBuffer, requiredAgreement, imMax.x, imMin.x, secondMin, secondMax, imageSize);
	}
	
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_PopulateHistogram: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif

	//normalize histogram
	threads = dim3(NUMTHREADS,1,1);
	grid = dim3( NUMTHREADS, 1, 1);
	float* dev_workingBuffer = 0;
	hipMalloc( &dev_workingBuffer, NUMTHREADS*NUMTHREADS*sizeof(float) );
	CopyBuffers<<<grid, threads, 0, *stream>>>(dev_workingBuffer, histogramGPU, NUMTHREADS*NUMTHREADS);
	float sum = 1.0f;
	for(int j = NUMTHREADS*NUMTHREADS / 2; j >= NUMTHREADS; j = j/2){
		dim3 tempGrid( j>NUMTHREADS ? j/NUMTHREADS : 1, 1, 1);
		SumOverLargeBuffer<<<tempGrid, threads, 0, *stream>>>(dev_workingBuffer,j,NUMTHREADS*NUMTHREADS);
	}
	SumData( NUMTHREADS, NUMTHREADS, 1, dev_workingBuffer, stream );
	hipMemcpyAsync( &sum, dev_workingBuffer, sizeof(float), hipMemcpyDeviceToHost, *stream );
	hipStreamSynchronize(*stream);
	hipFree(dev_workingBuffer);
	TranslateBuffer<<<grid,threads,0,*stream>>>(histogramGPU, 1.0f/sum, 0.0f, NUMTHREADS*NUMTHREADS);
	
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_NormalizeHistogram: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
	
	grid = GetGrid(imageSize);;
	for(int comp = 0; comp < NUMTHREADS; comp++){
		float secondMin = imMin.y + (float) comp * (imMax.y-imMin.y) / (float) NUMTHREADS;
		float secondMax = (comp != NUMTHREADS-1) ? imMin.y + (float) (comp+1) * (imMax.y-imMin.y) / (float) NUMTHREADS : FLT_MAX;
		kern_PopulateOutput2D<T><<<grid,threads,0,*stream>>>(histogramGPU+comp*NUMTHREADS, GPUOutputBuffer, GPUInputBuffer, imMax.x, imMin.x, secondMin, secondMax, imageSize);
	}

	hipMemcpyAsync( outputBuffer, GPUOutputBuffer, sizeof(float)*imageSize, hipMemcpyDeviceToHost, *stream );
	
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_CalculateTerms: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif

	hipFree(GPUOutputBuffer);
	hipFree(GPUInputBuffer);
	hipFree(GPUWorkingBuffer);
	hipFree(histogramGPU);
	hipFree(agreement);
	

}
