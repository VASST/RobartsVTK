#include "hip/hip_runtime.h"
#include "CUDA_loglikelihoodterm.h"
#include "stdio.h"
#include "hip/hip_runtime.h"
#include "float.h"
#include "limits.h"

#define NUMTHREADS 512

//#define DEBUG_VTKCUDA_ILLT

template<class T>
__global__ void kern_IncrementBuffer(T* labelBuffer, T desiredLabel, short* agreement, int size){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	short newAgreement = agreement[idx];
	T labelValue = labelBuffer[idx];
	newAgreement += (labelValue == desiredLabel) ? 1 : 0;
	if( idx < size ) agreement[idx] = newAgreement;
}

template void CUDA_ILLT_IncrementInformation<float>(float* labelData, float desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<double>(double* labelData, double desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<long>(long* labelData, long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<unsigned long>(unsigned long* labelData, unsigned long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<long long>(long long* labelData, long long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<unsigned long long>(unsigned long long* labelData, unsigned long long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<int>(int* labelData, int desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<unsigned int>(unsigned int* labelData, unsigned int desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<short>(short* labelData, short desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<unsigned short>(unsigned short* labelData, unsigned short desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<char>(char* labelData, char desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<signed char>(signed char* labelData, signed char desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_ILLT_IncrementInformation<unsigned char>(unsigned char* labelData, unsigned char desiredValue, short* agreement, int size, hipStream_t* stream);

template< class T >
void CUDA_ILLT_IncrementInformation(T* labelData, T desiredValue, short* agreement, int size, hipStream_t* stream){
    T* GPUBuffer = 0;

	hipMalloc((void**) &GPUBuffer, sizeof(T)*size);
	hipMemcpyAsync( GPUBuffer, labelData, sizeof(T)*size, hipMemcpyHostToDevice, *stream );

	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_IncrementInformation: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif

	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	kern_IncrementBuffer<T><<<grid,threads,0,*stream>>>(GPUBuffer, desiredValue, agreement, size);
	hipFree(GPUBuffer);

	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_IncrementInformation: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

template<class T>
__global__ void kern_ZeroOutBuffer(T* buffer, int size){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if( idx < size ) buffer[idx] = (T) 0;
}

void CUDA_ILLT_GetRelevantBuffers(short** agreement, float** output, int size, hipStream_t* stream){
	hipMalloc((void**) agreement, sizeof(short)*size);
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	kern_ZeroOutBuffer<short><<<grid,threads,0,*stream>>>(*agreement,size);
	hipMalloc((void**) output, sizeof(float)*size);

	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_GetRelevantBuffers: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

void CUDA_ILLT_CopyBackResult(float* GPUBuffer, float* CPUBuffer, int size, hipStream_t* stream){
	hipDeviceSynchronize();
	hipMemcpy( CPUBuffer, GPUBuffer, sizeof(float)*size, hipMemcpyDeviceToHost );
	hipFree(GPUBuffer);

	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_CopyBackResult: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

void CUDA_ILLT_AllocateHistogram(float** histogramGPU, int size, hipStream_t* stream){
	hipMalloc((void**) histogramGPU, sizeof(float)*size);
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_AllocateHistogram: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

void CUDA_ILLT_ReturnBuffer(float* buffer){
	hipFree(buffer);
}

template void CUDA_ILLT_CalculateHistogramAndTerms<double>(float* outputBuffer, float* histogramGPU, short* agreement, double* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<long>(float* outputBuffer, float* histogramGPU, short* agreement, long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<unsigned long>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<long long>(float* outputBuffer, float* histogramGPU, short* agreement, long long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<unsigned long long>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned long long* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<int>(float* outputBuffer, float* histogramGPU, short* agreement, int* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<unsigned int>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned int* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<short>(float* outputBuffer, float* histogramGPU, short* agreement, short* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<unsigned short>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned short* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<char>(float* outputBuffer, float* histogramGPU, short* agreement, char* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<signed char>(float* outputBuffer, float* histogramGPU, short* agreement, signed char* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<unsigned char>(float* outputBuffer, float* histogramGPU, short* agreement, unsigned char* image,  short requiredAgreement, int imageSize, hipStream_t* stream);
template void CUDA_ILLT_CalculateHistogramAndTerms<float>(float* outputBuffer, float* histogramGPU, short* agreement, float* image,	 short requiredAgreement, int imageSize, hipStream_t* stream);

template<class T>
__global__ void kern_PopulateWorkingUp(float* working, short* agreement, T* image, short requiredAgreement, int imageSize){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	float inputValue = (float) image[idx];
	short lAgreement = agreement[idx];
	float outputValue = (lAgreement < requiredAgreement) ? FLT_MIN: inputValue;
	if(idx < imageSize) working[idx] = outputValue;
}

template<class T>
__global__ void kern_PopulateWorkingDown(float* working, short* agreement, T* image, short requiredAgreement, int imageSize){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	float inputValue = (float) image[idx];
	short lAgreement = agreement[idx];
	float outputValue = (lAgreement < requiredAgreement) ? FLT_MAX: inputValue;
	if(idx < imageSize) working[idx] = outputValue;
}

__global__ void kern_PropogateUp(float* working, int span, int imageSize){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	float inputValue1 = working[idx];
	float inputValue2 = working[idx+span];
	float outputVal = (inputValue1 > inputValue2) ? inputValue1: inputValue2;
	if(idx+span < imageSize) working[idx] = outputVal;
}

__global__ void kern_PropogateDown(float* working, int span, int imageSize){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	float inputValue1 = working[idx];
	float inputValue2 = working[idx+span];
	float outputVal = (inputValue1 < inputValue2) ? inputValue1: inputValue2;
	if(idx+span < imageSize) working[idx] = outputVal;
}

template<class T>
__global__ void kern_PopulateHisto(float* histogramGPU, short* agreement, T* image, short requiredAgreement, float imMin, float imMax, int imageSize){
	__shared__ float histogram[NUMTHREADS];
	int idx = threadIdx.x;
	imMin -= (imMax-imMin)*0.00625;
	imMax += (imMax-imMin)*0.00625;

	histogram[idx] = 1e-10f;
	__syncthreads();
	int repetitions = (imageSize-1) / blockDim.x + 1;
	int idxCurr = idx;
	for(int i = 0; i < repetitions; i++, idxCurr += blockDim.x){
		short localAgreement = agreement[idxCurr];
		float localValue = (float) image[idxCurr];
		int histInPos = (int) ( (float) (NUMTHREADS-1) * ((localValue-imMin) / (imMax-imMin)) + 0.5f );
		int histPos = idx;
		for(int h = 0; h < NUMTHREADS; h++){
			__syncthreads();
			histogram[histPos] += (idxCurr < imageSize && localAgreement >= requiredAgreement && histPos == histInPos) ? 1 : 0;
			histPos += (histPos < NUMTHREADS-1) ? 1: -histPos;	
		}
	}
	__syncthreads();

	//normalize inefficiently
	if(idx==0){
		float sum = 0.0f;
		for(int h = 0; h < NUMTHREADS; h++)
			sum += histogram[h];
		for(int h = 0; h < NUMTHREADS; h++)
			histogram[h] /= sum;
	}
	
	__syncthreads();
	histogramGPU[idx] = histogram[idx];

}

template<class T>
__global__ void kern_PopulateOutput(float* histogramGPU, float* output, T* image, float imMin, float imMax, int imageSize){
	__shared__ float histogram[NUMTHREADS];
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	imMin -= (imMax-imMin)*0.00625;
	imMax += (imMax-imMin)*0.00625;
	if( threadIdx.x < NUMTHREADS ) histogram[threadIdx.x] = histogramGPU[threadIdx.x];
	__syncthreads();
	
	float localValue = (float) image[idx];
	int histPos = (int) ( (float) (NUMTHREADS-1) * ((localValue-imMin) / (imMax-imMin)) + 0.5f );
	float histVal = (histPos < NUMTHREADS && histPos >= 0) ? histogram[histPos] : 1e-10f;
	histVal = (histVal < 1e-10f) ? 1e-10f : histVal;
	histVal = log(histVal) / log(1e-10f);
	if(idx < imageSize) output[idx] = histVal;

}

template< class T >
void CUDA_ILLT_CalculateHistogramAndTerms(float* outputBuffer, float* histogramGPU, short* agreement, T* image, short requiredAgreement, int imageSize, hipStream_t* stream){
	
	T* GPUInputBuffer = 0;
	float* GPUOutputBuffer = 0;
	float* GPUWorkingBuffer = 0;
	hipMalloc((void**) &GPUInputBuffer, sizeof(T)*imageSize);
	hipMalloc((void**) &GPUOutputBuffer, sizeof(float)*imageSize);
	hipMalloc((void**) &GPUWorkingBuffer, sizeof(float)*imageSize);
	hipMemcpyAsync( GPUInputBuffer, image, sizeof(T)*imageSize, hipMemcpyHostToDevice, *stream );

	float imMax = 0;
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (imageSize-1)/NUMTHREADS+1, 1, 1);
	kern_PopulateWorkingUp<T><<<grid,threads,0,*stream>>>(GPUWorkingBuffer, agreement, GPUInputBuffer, requiredAgreement, imageSize);
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_CalculateMinMax: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
	for(int t = (imageSize-1)/2+1; t > 0; t/=2){
		threads = dim3(NUMTHREADS,1,1);
		grid = dim3( (t-1)/NUMTHREADS+1, 1, 1);
		kern_PropogateUp<<<grid,threads,0,*stream>>>(GPUWorkingBuffer, t, imageSize);

		
		#ifdef DEBUG_VTKCUDA_ILLT
			hipDeviceSynchronize();
			printf( "CUDA_ILLT_CalculateMinMax: " );
			printf( hipGetErrorString( hipGetLastError() ) );
			printf( "\n" );
		#endif
	}
	hipMemcpyAsync( &imMax, GPUWorkingBuffer, sizeof(float), hipMemcpyDeviceToHost, *stream );
	hipDeviceSynchronize();

	float imMin = 0;
	threads = dim3(NUMTHREADS,1,1);
	grid = dim3( (imageSize-1)/NUMTHREADS+1, 1, 1);
	kern_PopulateWorkingDown<T><<<grid,threads,0,*stream>>>(GPUWorkingBuffer, agreement, GPUInputBuffer, requiredAgreement, imageSize);
	for(int t = (imageSize-1)/2+1; t > 0; t/=2){
		threads = dim3(NUMTHREADS,1,1);
		grid = dim3( (t-1)/NUMTHREADS+1, 1, 1);
		kern_PropogateDown<<<grid,threads,0,*stream>>>(GPUWorkingBuffer, t, imageSize);

		
		#ifdef DEBUG_VTKCUDA_ILLT
			hipDeviceSynchronize();
			printf( "CUDA_ILLT_CalculateMinMax: " );
			printf( hipGetErrorString( hipGetLastError() ) );
			printf( "\n" );
		#endif
	}
	hipMemcpyAsync( &imMin, GPUWorkingBuffer, sizeof(float), hipMemcpyDeviceToHost, *stream );
	hipDeviceSynchronize();
	

	threads = dim3(NUMTHREADS,1,1);
	grid = dim3( 1, 1, 1);
	kern_PopulateHisto<T><<<grid,threads,0,*stream>>>(histogramGPU, agreement, GPUInputBuffer, requiredAgreement, imMax, imMin, imageSize);
	
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_CalculateHistogram: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif


	grid = dim3( (imageSize-1)/NUMTHREADS+1, 1, 1);
	kern_PopulateOutput<T><<<grid,threads,0,*stream>>>(histogramGPU, GPUOutputBuffer, GPUInputBuffer, imMax, imMin, imageSize);

	hipMemcpyAsync( outputBuffer, GPUOutputBuffer, sizeof(float)*imageSize, hipMemcpyDeviceToHost, *stream );
	
	#ifdef DEBUG_VTKCUDA_ILLT
		hipDeviceSynchronize();
		printf( "CUDA_ILLT_CalculateTerms: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif

	hipFree(GPUOutputBuffer);
	hipFree(GPUInputBuffer);
	hipFree(GPUWorkingBuffer);
	hipFree(histogramGPU);
	

}
