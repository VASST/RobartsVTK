#include "hip/hip_runtime.h"
#include "CUDA_kohonenreprojector.h"
#include "CUDA_commonKernels.h"
#include <float.h>

__constant__ Kohonen_Reprojection_Information info;
texture<float, 3, hipReadModeElementType> Kohonen_Map;

__global__ void ApplyReprojection(float2* InputBuffer, float* OutputBuffer){

	//shared memory
	__shared__ float2 InputIndices[NUMTHREADS];

	//get volume and output dimensions
	int outBufferSize = info.VolumeSize[0]*info.VolumeSize[1]*info.VolumeSize[2]*info.NumberOfDimensions;

	//find index in current block
	int individualInputIndex = CUDASTDOFFSET;

	//fetch indexing information from input buffer into shared memory
	InputIndices[threadIdx.x] = InputBuffer[individualInputIndex];
	__syncthreads();

	//find the starting ouput index coallesced for output
	int individualOutputIndex = threadIdx.x + NUMTHREADS * blockDim.x * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z));

	//for each component
	for( int i = 0; i < info.NumberOfDimensions; i++ ){

		//fetch input index from shared memory
		int currInputIndex = (i*NUMTHREADS + threadIdx.x) / info.NumberOfDimensions;
		float2 currLoc = InputIndices[currInputIndex];
		int currComponent = 2*((i*NUMTHREADS + threadIdx.x) % info.NumberOfDimensions)+1;

		//fetch information from texture
		float reprojValue = tex3D(Kohonen_Map, (float) currComponent, currLoc.x, currLoc.y);

		//write to output
		if( individualOutputIndex < outBufferSize )
			OutputBuffer[individualOutputIndex] = reprojValue;

		//find new output index for next iteration, coallesced for output
		individualOutputIndex += NUMTHREADS;

	}
}

void CUDAalgo_reprojectKohonenMap( float* inputData, float* inputKohonen, float* outputData,
								Kohonen_Reprojection_Information& information,
								hipStream_t* stream ){

	//copy information to GPU
	hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &information, sizeof(Kohonen_Reprojection_Information) );

	//allocate output image and load the input indices into the device
	int s = information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2];
	float* dev_OutputImage = 0;
	hipMalloc( &dev_OutputImage, sizeof(float)*s*information.NumberOfDimensions );
	float2* dev_InputImage = 0;
	hipMalloc( &dev_InputImage, sizeof(float2)*s );
	hipMemcpyAsync(dev_InputImage, inputData, sizeof(float2)*s, hipMemcpyHostToDevice, *stream);

	//create device memory for the map and load Kohonen map into device memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* dev_KSOM = 0;
	hipExtent KSOMSize;
	KSOMSize.width = 2*information.NumberOfDimensions+1;
	KSOMSize.height = information.KohonenMapSize[0];
	KSOMSize.depth = information.KohonenMapSize[1];
	hipMalloc3DArray(&dev_KSOM, &channelDesc, KSOMSize);
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr( (void*) inputKohonen, KSOMSize.width*sizeof(float),
												KSOMSize.width, KSOMSize.height);
	copyParams.dstArray = dev_KSOM;
	copyParams.extent   = KSOMSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	hipMemcpy3DAsync(&copyParams, *stream);
	hipStreamSynchronize(*stream);

	//bind KSOM array to 3D texture
	Kohonen_Map.normalized = false; // access with unnormalized texture coordinates
	Kohonen_Map.filterMode = hipFilterModePoint;
	Kohonen_Map.addressMode[0] = hipAddressModeClamp;
	Kohonen_Map.addressMode[1] = hipAddressModeClamp;
	Kohonen_Map.addressMode[2] = hipAddressModeClamp;
	hipBindTextureToArray(Kohonen_Map, dev_KSOM, channelDesc);

	//translate input indices
	dim3 grid = GetGrid(information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2]);
	dim3 threads(NUMTHREADS,1,1);
	hipDeviceSynchronize();
	ApplyReprojection<<<grid, threads, 0, *stream >>>(dev_InputImage, dev_OutputImage);
	hipDeviceSynchronize();

	//deallocate the device memory for the Kohonen map
	hipUnbindTexture(Kohonen_Map);
	hipFreeArray(dev_KSOM);

	//deallocate the memory for the input indices
	hipFree(dev_InputImage);

	//copy out the output values and deallocate remaining device mempory
	hipMemcpyAsync(outputData, dev_OutputImage, sizeof(float)*s*information.NumberOfDimensions, hipMemcpyDeviceToHost, *stream);
	hipFree( dev_OutputImage );
	hipStreamSynchronize(*stream);
}