#include "CUDA_kohonenreprojector.h"
#include <float.h>

__constant__ Kohonen_Reprojection_Information info;
texture<float, 3, hipReadModeElementType> Kohonen_Map;

void CUDAalgo_reprojectKohonenMap( float* inputData, float* inputKohonen, short* outputData,
								Kohonen_Reprojection_Information& information,
								hipStream_t* stream ){

	//copy information to GPU
	hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &information, sizeof(Kohonen_Reprojection_Information) );

	//allocate output image and load the input indices into the device
	float* dev_OutputImage = 0;
	int s = information.NumberOfDimensions*information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2];
	hipMalloc( &dev_OutputImage, sizeof(float)*s );

	//create device memory for the map and load Kohonen map into device memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* dev_KSOM = 0;
	hipExtent KSOMSize;
	KSOMSize.width = information.NumberOfDimensions;
	KSOMSize.height = information.KohonenMapSize[0];
	KSOMSize.depth = information.KohonenMapSize[1];
	hipMalloc3DArray(&dev_KSOM, &channelDesc, KSOMSize);
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr( (void*) inputKohonen, KSOMSize.width*sizeof(float),
												KSOMSize.width, KSOMSize.height);
	copyParams.dstArray = dev_KSOM;
	copyParams.extent   = KSOMSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	hipMemcpy3DAsync(&copyParams, *stream);

	//bind KSOM array to 3D texture
	Kohonen_Map.normalized = false; // access with unnormalized texture coordinates
	Kohonen_Map.filterMode = hipFilterModeLinear;
	Kohonen_Map.addressMode[0] = hipAddressModeClamp;
	Kohonen_Map.addressMode[1] = hipAddressModeClamp;
	Kohonen_Map.addressMode[2] = hipAddressModeClamp;
	hipBindTextureToArray(Kohonen_Map, dev_KSOM, channelDesc);

	//translate input indices

	//deallocate the device memory for the Kohonen map

	//deallocate the memory for the input indices

	//copy out the output values and deallocate remaining device mempory
}