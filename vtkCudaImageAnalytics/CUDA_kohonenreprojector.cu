#include "hip/hip_runtime.h"
#include "CUDA_kohonenreprojector.h"
#include <float.h>

__constant__ Kohonen_Reprojection_Information info;
texture<float, 3, hipReadModeElementType> Kohonen_Map;

#define NUM_THREADS 256

__global__ void ApplyReprojection(float2* InputBuffer, float* OutputBuffer){

	//shared memory
	__shared__ float2 InputIndices[NUM_THREADS];

	//get volume and output dimensions
	int outBufferSize = info.VolumeSize[0]*info.VolumeSize[1]*info.VolumeSize[2]*info.NumberOfDimensions;

	//find index in current block
	int individualInputIndex = blockIdx.x*NUM_THREADS + threadIdx.x;

	//fetch indexing information from input buffer into shared memory
	InputIndices[threadIdx.x] = InputBuffer[individualInputIndex];
	__syncthreads();

	//find the starting ouput index coallesced for output
	int individualOutputIndex = blockIdx.x*NUM_THREADS*info.NumberOfDimensions + threadIdx.x;

	//for each component
	for( int i = 0; i < info.NumberOfDimensions; i++ ){

		//fetch input index from shared memory
		int currInputIndex = (i*NUM_THREADS + threadIdx.x) / info.NumberOfDimensions;
		float2 currLoc = InputIndices[currInputIndex];
		int currComponent = 2*((i*NUM_THREADS + threadIdx.x) % info.NumberOfDimensions);

		//fetch information from texture
		float reprojValue = tex3D(Kohonen_Map, (float) currComponent, currLoc.x, currLoc.y);

		//write to output
		if( individualOutputIndex < outBufferSize )
			OutputBuffer[individualOutputIndex] = reprojValue;

		//find new output index for next iteration, coallesced for output
		individualOutputIndex += NUM_THREADS;

	}
}

void CUDAalgo_reprojectKohonenMap( float* inputData, float* inputKohonen, float* outputData,
								Kohonen_Reprojection_Information& information,
								hipStream_t* stream ){

	//copy information to GPU
	hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &information, sizeof(Kohonen_Reprojection_Information) );

	//allocate output image and load the input indices into the device
	int s = information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2];
	float* dev_OutputImage = 0;
	hipMalloc( &dev_OutputImage, sizeof(float)*s*information.NumberOfDimensions );
	float2* dev_InputImage = 0;
	hipMalloc( &dev_InputImage, sizeof(float2)*s );
	hipMemcpyAsync(dev_InputImage, inputData, sizeof(float2)*s, hipMemcpyHostToDevice, *stream);

	//create device memory for the map and load Kohonen map into device memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* dev_KSOM = 0;
	hipExtent KSOMSize;
	KSOMSize.width = 2*information.NumberOfDimensions;
	KSOMSize.height = information.KohonenMapSize[0];
	KSOMSize.depth = information.KohonenMapSize[1];
	hipMalloc3DArray(&dev_KSOM, &channelDesc, KSOMSize);
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr( (void*) inputKohonen, KSOMSize.width*sizeof(float),
												KSOMSize.width, KSOMSize.height);
	copyParams.dstArray = dev_KSOM;
	copyParams.extent   = KSOMSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	hipMemcpy3DAsync(&copyParams, *stream);
	hipStreamSynchronize(*stream);

	//bind KSOM array to 3D texture
	Kohonen_Map.normalized = false; // access with unnormalized texture coordinates
	Kohonen_Map.filterMode = hipFilterModePoint;
	Kohonen_Map.addressMode[0] = hipAddressModeClamp;
	Kohonen_Map.addressMode[1] = hipAddressModeClamp;
	Kohonen_Map.addressMode[2] = hipAddressModeClamp;
	hipBindTextureToArray(Kohonen_Map, dev_KSOM, channelDesc);

	//translate input indices
	dim3 grid((information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2] + NUM_THREADS-1) / NUM_THREADS,1,1);
	dim3 threads(NUM_THREADS,1,1);
	hipDeviceSynchronize();
	ApplyReprojection<<<grid, threads, 0, *stream >>>(dev_InputImage, dev_OutputImage);
	hipDeviceSynchronize();

	//deallocate the device memory for the Kohonen map
	hipUnbindTexture(Kohonen_Map);
	hipFreeArray(dev_KSOM);

	//deallocate the memory for the input indices
	hipFree(dev_InputImage);

	//copy out the output values and deallocate remaining device mempory
	hipMemcpyAsync(outputData, dev_OutputImage, sizeof(float)*s*information.NumberOfDimensions, hipMemcpyDeviceToHost, *stream);
	hipFree( dev_OutputImage );
	hipStreamSynchronize(*stream);
}