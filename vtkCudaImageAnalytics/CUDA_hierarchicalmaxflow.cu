#include "hip/hip_runtime.h"
#include "CUDA_hierarchicalmaxflow.h"
#include "stdio.h"
#include "hip/hip_runtime.h"

#define NUMTHREADS 512

int CUDA_GetGPUBuffers( int maxNumber, float** buffer, int volSize ){

	size_t freeMemory, totalMemory;
	hipError_t nErr = hipSuccess;
	hipMemGetInfo(&freeMemory, &totalMemory);

    printf("===========================================================\n");
    printf("Free/Total(kB): %f/%f\n", (float)freeMemory/1024.0f, (float)totalMemory/1024.0f);

	while( maxNumber > 0 ){
		nErr = hipMalloc((void**) buffer, sizeof(float)*maxNumber*volSize);
		if( nErr == hipSuccess ) break;
		maxNumber--; 
	}
	
	hipMemGetInfo(&freeMemory, &totalMemory);
    printf("===========================================================\n");
    printf("Free/Total(kB): %f/%f\n", (float)freeMemory/1024.0f, (float)totalMemory/1024.0f);

	return maxNumber;

}

void CUDA_ReturnGPUBuffers(float* buffer){
	hipFree(buffer);
}


void CUDA_CopyBufferToCPU(float* GPUBuffer, float* CPUBuffer, int size, hipStream_t* stream){
	hipMemcpyAsync( CPUBuffer, GPUBuffer, sizeof(float)*size, hipMemcpyDeviceToHost, *stream );
}

void CUDA_CopyBufferToGPU(float* GPUBuffer, float* CPUBuffer, int size, hipStream_t* stream){
	hipMemcpyAsync( GPUBuffer, CPUBuffer, sizeof(float)*size, hipMemcpyHostToDevice, *stream );
}

__global__ void kern_ZeroOutBuffer(float* buffer, int size){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if( idx < size ) buffer[idx] = 0.0f;
}

void CUDA_zeroOutBuffer(float* GPUBuffer, int size, hipStream_t* stream){
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	kern_ZeroOutBuffer<<<grid,threads,0,*stream>>>(GPUBuffer,size);
}

__global__ void kern_DivideAndStoreBuffer(float* inBuffer, float* outBuffer, float number, int size){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	float value = inBuffer[idx] * number;
	if( idx < size ) outBuffer[idx] = value;
}

void CUDA_divideAndStoreBuffer(float* inBuffer, float* outBuffer, float number, int size, hipStream_t* stream){
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	kern_DivideAndStoreBuffer<<<grid,threads,0,*stream>>>(inBuffer,outBuffer,1.0f/number,size);
}