#include "hip/hip_runtime.h"
#include "CUDA_cttoussimulation.h"

//3D input data (read-only texture with corresponding opague device memory back)
texture<float, 3, hipReadModeElementType> ct_input_texture;
hipArray* ct_input_array = 0;

//parameters held in constant memory
__constant__ CT_To_US_Information info;

//device code to collect information on each of the sample points
__device__ void CUDAkernel_CollectSamples(	int2 index,
						float3& rayStart,
						float3& rayInc,
						float* outputDensity,
						float* outputTransmission,
						float* outputReflection,
						uchar3* outputUltrasound){

	//collect parameters from constant memory
	uint3 volumeSize = info.VolumeSize;
	float3 spacing = info.spacing;
	unsigned int numStepsToTake = info.Resolution.z;
	unsigned int xResolution = info.Resolution.x;
	unsigned int yResolution = info.Resolution.y;
	unsigned int actIndex = index.x + index.y* xResolution;
	unsigned int indexInc = xResolution*yResolution;
	bool isValid = (index.y < yResolution);
	float threshold = info.reflectionThreshold;
	float densitySlope = 0.5f*0.00025f;
	float densityIntercept = 512.0f*0.00025f;
	float HounsFieldScale = info.hounsfieldScale;
	float HounsFieldOffset = info.hounsfieldOffset;
	float alpha = info.alpha;
	float beta = info.beta;
	float bias = info.bias;

	float directionMag = sqrt( rayInc.x*rayInc.x + rayInc.y*rayInc.y + rayInc.z*rayInc.z );
	float worldDirectionMag = 2.0f * sqrt( rayInc.x*rayInc.x/(spacing.x*spacing.x) +
										   rayInc.y*rayInc.y/(spacing.y*spacing.y) +
										   rayInc.z*rayInc.z/(spacing.z*spacing.z) );

	//set up running accumulators
	float transmission = 1.0f;

	//set up output scaling parameters
	float multiplier = info.a;
	float divisor = 1.0f / log(1.0f+multiplier);

	for(unsigned int numStepsTaken = 0; numStepsTaken < numStepsToTake; numStepsTaken++){

		//create default values for the sample point
		float density = 0.0f;
		float transmissionLost = 1.0f;
		float pointReflection = 0.0f;

		float attenuation = 0.0f;

		__syncthreads();
		if(!(rayStart.x < 0.0f || rayStart.y < 0.0f || rayStart.z < 0.0f ||
			 rayStart.x > (float)(volumeSize.x - 1) ||
			 rayStart.y > (float)(volumeSize.y - 1) ||
			 rayStart.y > (float)(volumeSize.y - 1) )){

			//get the attenuation and gradient of the attenuation in Hounsfield units
			attenuation = HounsFieldScale*tex3D(ct_input_texture, rayStart.x, rayStart.y, rayStart.z) + HounsFieldOffset;
			float gradientX = HounsFieldScale*(tex3D(ct_input_texture, rayStart.x+1.0f, rayStart.y, rayStart.z) - tex3D(ct_input_texture, rayStart.x-1.0f, rayStart.y, rayStart.z)) * spacing.x;
			float gradientY = HounsFieldScale*(tex3D(ct_input_texture, rayStart.x, rayStart.y+1.0f, rayStart.z) - tex3D(ct_input_texture, rayStart.x, rayStart.y-1.0f, rayStart.z)) * spacing.y;
			float gradientZ = HounsFieldScale*(tex3D(ct_input_texture, rayStart.x, rayStart.y, rayStart.z+1.0f) - tex3D(ct_input_texture, rayStart.x, rayStart.y, rayStart.z-1.0f)) * spacing.z;
			float gradMagSquared = gradientX*gradientX + gradientY*gradientY + gradientZ*gradientZ;
			float gradMag = sqrt( gradMagSquared );

			//calculate the reflection, density and transmission at this sample point
			transmissionLost = (gradMag < threshold) ? saturate( 1.0f - gradMagSquared * worldDirectionMag / (4.0f * attenuation * attenuation) ) : 0.0f;
			pointReflection  = transmission * (rayInc.x*gradientX + rayInc.y*gradientY + rayInc.z*gradientZ) * gradMag / ( 4.0f * attenuation * attenuation * directionMag );
			density          = (transmission > 0.0f) ? densitySlope * attenuation + densityIntercept : 0.0f;

		}

		//scale the point reflection
		pointReflection = saturate( log( 1 + multiplier * pointReflection ) * divisor );
		
		//output the reflection and density
		__syncthreads();
		if( isValid ) outputReflection[actIndex] = pointReflection;
		__syncthreads();
		if( isValid ) outputDensity[actIndex] = density;
		__syncthreads();
		
		//create the output image
		uchar3 outputImage;
		outputImage.x = 255.0f * saturate(alpha*density+beta*pointReflection+bias);
		outputImage.y = 255.0f * saturate(alpha*density+beta*pointReflection+bias);
		outputImage.z = 255.0f * saturate(alpha*density+beta*pointReflection+bias);

		//output the simulated ultrasound
		__syncthreads();
		if( isValid ) outputUltrasound[actIndex] = outputImage;
		__syncthreads();

		//update the running values
		transmission *= transmissionLost;

		//output the transmission
		__syncthreads();
		if( isValid ) outputTransmission[actIndex] = transmission;
		__syncthreads();

		//update the sampling location
		actIndex += indexInc;
		rayStart.x += rayInc.x;
		rayStart.y += rayInc.y;
		rayStart.z += rayInc.z;

	}


}

//device code to determine from the parameters, the start, end and increment vectors in volume space
__device__ void CUDAkernel_FindVectors(	float2 nIndex,
										float3& rayStart,
										float3& rayInc){

	
	//find the US coordinates of this particular beam's Start point
	float3 usStart;
	usStart.x = tan( info.fanAngle.x * nIndex.x );
	usStart.y = tan( info.fanAngle.y * nIndex.y );
	usStart.z = __fsqrt_rz( info.StartDepth * info.StartDepth / 
							( 1.0f + usStart.x*usStart.x + usStart.y*usStart.y) );
	usStart.x = 0.5f * info.probeWidth.x * nIndex.x + usStart.x*usStart.z;
	usStart.y = 0.5f * info.probeWidth.y * nIndex.y + usStart.y*usStart.z;
	__syncthreads();

	//find the Start vector in world coordinates
	float4 worldStart;
	worldStart.x = info.UltraSoundToWorld[ 0] * usStart.x + info.UltraSoundToWorld[ 1] * usStart.y + info.UltraSoundToWorld[ 2] * usStart.z + info.UltraSoundToWorld[ 3];
	worldStart.y = info.UltraSoundToWorld[ 4] * usStart.x + info.UltraSoundToWorld[ 5] * usStart.y + info.UltraSoundToWorld[ 6] * usStart.z + info.UltraSoundToWorld[ 7];
	worldStart.z = info.UltraSoundToWorld[ 8] * usStart.x + info.UltraSoundToWorld[ 9] * usStart.y + info.UltraSoundToWorld[10] * usStart.z + info.UltraSoundToWorld[11];
	worldStart.w = info.UltraSoundToWorld[12] * usStart.x + info.UltraSoundToWorld[13] * usStart.y + info.UltraSoundToWorld[14] * usStart.z + info.UltraSoundToWorld[15];
	__syncthreads();
	worldStart.x /= worldStart.w; 
	worldStart.y /= worldStart.w; 
	worldStart.z /= worldStart.w;

	//transform the Start into volume co-ordinates
	__syncthreads();
	rayStart.x   = info.WorldToVolume[ 0]*worldStart.x + info.WorldToVolume[ 1]*worldStart.y + info.WorldToVolume[ 2]*worldStart.z + info.WorldToVolume[ 3];
	rayStart.y   = info.WorldToVolume[ 4]*worldStart.x + info.WorldToVolume[ 5]*worldStart.y + info.WorldToVolume[ 6]*worldStart.z + info.WorldToVolume[ 7];
	rayStart.z   = info.WorldToVolume[ 8]*worldStart.x + info.WorldToVolume[ 9]*worldStart.y + info.WorldToVolume[10]*worldStart.z + info.WorldToVolume[11];
	worldStart.w = info.WorldToVolume[12]*worldStart.x + info.WorldToVolume[13]*worldStart.y + info.WorldToVolume[14]*worldStart.z + info.WorldToVolume[15];
	__syncthreads();
	rayStart.x /= worldStart.w;
	rayStart.y /= worldStart.w;
	rayStart.z /= worldStart.w;
	

	//find the US coordinates of this particular beam's Start point
	float3 usEnd;
	usEnd.x = tan( info.fanAngle.x * nIndex.x );
	usEnd.y = tan( info.fanAngle.y * nIndex.y );
	usEnd.z = __fsqrt_rz( (info.EndDepth * info.EndDepth) / 
						( 1.0f + usEnd.x*usEnd.x + usEnd.y*usEnd.y) );
	usEnd.x = 0.5f * info.probeWidth.x * nIndex.x + usEnd.x*usEnd.z;
	usEnd.y = 0.5f * info.probeWidth.y * nIndex.y + usEnd.y*usEnd.z;
	
	//find the End vector in world coordinates
	float4 worldEnd;
	__syncthreads();
	worldEnd.x = info.UltraSoundToWorld[ 0] * usEnd.x + info.UltraSoundToWorld[ 1] * usEnd.y + info.UltraSoundToWorld[ 2] * usEnd.z + info.UltraSoundToWorld[ 3];
	worldEnd.y = info.UltraSoundToWorld[ 4] * usEnd.x + info.UltraSoundToWorld[ 5] * usEnd.y + info.UltraSoundToWorld[ 6] * usEnd.z + info.UltraSoundToWorld[ 7];
	worldEnd.z = info.UltraSoundToWorld[ 8] * usEnd.x + info.UltraSoundToWorld[ 9] * usEnd.y + info.UltraSoundToWorld[10] * usEnd.z + info.UltraSoundToWorld[11];
	worldEnd.w = info.UltraSoundToWorld[12] * usEnd.x + info.UltraSoundToWorld[13] * usEnd.y + info.UltraSoundToWorld[14] * usEnd.z + info.UltraSoundToWorld[15];
	__syncthreads();
	worldEnd.x /= worldEnd.w; 
	worldEnd.y /= worldEnd.w; 
	worldEnd.z /= worldEnd.w;

	//transform the End into volume co-ordinates
	float3 rayEnd;
	__syncthreads();
	rayEnd.x   = info.WorldToVolume[ 0]*worldEnd.x + info.WorldToVolume[ 1]*worldEnd.y + info.WorldToVolume[ 2]*worldEnd.z + info.WorldToVolume[ 3];
	rayEnd.y   = info.WorldToVolume[ 4]*worldEnd.x + info.WorldToVolume[ 5]*worldEnd.y + info.WorldToVolume[ 6]*worldEnd.z + info.WorldToVolume[ 7];
	rayEnd.z   = info.WorldToVolume[ 8]*worldEnd.x + info.WorldToVolume[ 9]*worldEnd.y + info.WorldToVolume[10]*worldEnd.z + info.WorldToVolume[11];
	worldEnd.w = info.WorldToVolume[12]*worldEnd.x + info.WorldToVolume[13]*worldEnd.y + info.WorldToVolume[14]*worldEnd.z + info.WorldToVolume[15];
	__syncthreads();
	rayEnd.x /= worldEnd.w;
	rayEnd.y /= worldEnd.w;
	rayEnd.z /= worldEnd.w;

	//calculate the increment vector
	__syncthreads();
	rayInc.x = (rayStart.x-rayEnd.x) / info.Resolution.z;
	rayInc.y = (rayStart.y-rayEnd.y) / info.Resolution.z;
	rayInc.z = (rayStart.z-rayEnd.z) / info.Resolution.z;

}

__global__ void CUDAkernel_CreateSimulatedUS(	float* outputDensity,
												float* outputTransmission,
												float* outputReflection,
												uchar3* outputUltrasound){

	//find x index value in the simulated ultrasound image
	int2 index;
	index.x = (threadIdx.x + blockDim.x * blockIdx.x) % info.Resolution.x;
	index.y = (threadIdx.x + blockDim.x * blockIdx.x) / info.Resolution.x;

	//find the normalized indices
	float2 normIndex;
	normIndex.x = (float) (index.x+index.x) / (float) info.Resolution.x - 1.0f;
	normIndex.y = (float) (index.y+index.y) / (float) info.Resolution.y - 1.0f;
	
	//starting and increment vectors
	float3 rayStart;
	float3 rayInc;
	CUDAkernel_FindVectors( normIndex, rayStart, rayInc );

	//simulate the ultrasound (writing to the output buffers)
	CUDAkernel_CollectSamples( index, rayStart, rayInc, outputDensity, outputTransmission, outputReflection, outputUltrasound );

}

void CUDAsetup_loadCTImage( float* CTImage, CT_To_US_Information& information, hipStream_t* stream){

	//free the array if there is another image residing
	hipStreamSynchronize( *stream );
	if(ct_input_array ) hipFreeArray(ct_input_array );

	//find the volume size
	hipExtent volumeSize;
	volumeSize.width = information.VolumeSize.x;
	volumeSize.height = information.VolumeSize.y;
	volumeSize.depth = information.VolumeSize.z;

	// create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&ct_input_array, &channelDesc, volumeSize);

	// copy data to 3D array
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr(CTImage, volumeSize.width*sizeof(float), volumeSize.width, volumeSize.height);
	copyParams.dstArray = ct_input_array;
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	hipMemcpy3DAsync(&copyParams, *stream);

	// set the texture parameters
	ct_input_texture.normalized = false;						// access with unnormalized texture coordinates
	ct_input_texture.filterMode = hipFilterModeLinear;			// linear interpolation
	ct_input_texture.addressMode[0] = hipAddressModeClamp;		// wrap texture coordinates
	ct_input_texture.addressMode[1] = hipAddressModeClamp;
	ct_input_texture.addressMode[2] = hipAddressModeClamp;

	//bind the texture in
	hipBindTextureToArray(ct_input_texture, ct_input_array, channelDesc);

}

void CUDAsetup_unloadCTImage(hipStream_t* stream){
	hipStreamSynchronize( *stream );
	if(ct_input_array ) hipFreeArray(ct_input_array );
	ct_input_array = 0;
}

void CUDAalgo_simulateUltraSound(	float* outputDensity, float* outputTransmission, float* outputReflection, unsigned char* outputUltrasound,
									CT_To_US_Information& information, hipStream_t* stream ){

	//copy the information to the device
	hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &information, sizeof(CT_To_US_Information), 0, hipMemcpyHostToDevice, *stream);

	//allocate the device output buffers
	float* device_output_dens;
	float* device_output_trans;
	float* device_output_refl;
	uchar3* device_output_us;
	hipMalloc( (void**) &device_output_dens,  sizeof(float)*information.Resolution.x*information.Resolution.y*information.Resolution.z );
	hipMalloc( (void**) &device_output_trans, sizeof(float)*information.Resolution.x*information.Resolution.y*information.Resolution.z );
	hipMalloc( (void**) &device_output_refl,  sizeof(float)*information.Resolution.x*information.Resolution.y*information.Resolution.z );
	hipMalloc( (void**) &device_output_us,    3*sizeof(unsigned char)*information.Resolution.x*information.Resolution.y*information.Resolution.z );

	//simulate the ultrasound
	int maxBlockSize = 256;
	dim3 threads( maxBlockSize, 1, 1);
	int gridSize = information.Resolution.x * information.Resolution.y / maxBlockSize + ( (information.Resolution.x % maxBlockSize == 0 ) ? 0 : 1 );
	dim3 grid( gridSize, 1, 1);
	CUDAkernel_CreateSimulatedUS<<< grid, threads, 0, *stream >>>( device_output_dens, device_output_trans, device_output_refl, device_output_us );

	//copy the results
	hipMemcpyAsync( (void*) outputDensity,      (void*) device_output_dens,  sizeof(float)*information.Resolution.x*information.Resolution.y*information.Resolution.z, hipMemcpyDeviceToHost, *stream );
	hipMemcpyAsync( (void*) outputTransmission, (void*) device_output_trans, sizeof(float)*information.Resolution.x*information.Resolution.y*information.Resolution.z, hipMemcpyDeviceToHost, *stream );
	hipMemcpyAsync( (void*) outputReflection,   (void*) device_output_refl,  sizeof(float)*information.Resolution.x*information.Resolution.y*information.Resolution.z, hipMemcpyDeviceToHost, *stream );
	hipMemcpyAsync( (void*) outputUltrasound,   (void*) device_output_us,    3*sizeof(unsigned char)*information.Resolution.x*information.Resolution.y*information.Resolution.z, hipMemcpyDeviceToHost, *stream );
	hipStreamSynchronize( *stream );

	//free the device buffers
	hipFree(device_output_dens);
	hipFree(device_output_trans);
	hipFree(device_output_refl);
	hipFree(device_output_us);

}