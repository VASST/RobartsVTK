#include "hip/hip_runtime.h"
#include "CUDA_atlasprobability.h"
#include "stdio.h"
#include "hip/hip_runtime.h"

#define NUMTHREADS 512

//#define DEBUG_VTKCUDA_IALP

template<class T>
__global__ void kern_IncrementBuffer(T* labelBuffer, T desiredLabel, short* agreement, int size){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	short newAgreement = agreement[idx];
	T labelValue = labelBuffer[idx];
	newAgreement += (labelValue == desiredLabel) ? 1 : 0;
	if( idx < size ) agreement[idx] = newAgreement;
}

template void CUDA_IncrementInformation<float>(float* labelData, float desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<double>(double* labelData, double desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<long>(long* labelData, long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<unsigned long>(unsigned long* labelData, unsigned long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<long long>(long long* labelData, long long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<unsigned long long>(unsigned long long* labelData, unsigned long long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<int>(int* labelData, int desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<unsigned int>(unsigned int* labelData, unsigned int desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<short>(short* labelData, short desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<unsigned short>(unsigned short* labelData, unsigned short desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<char>(char* labelData, char desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<signed char>(signed char* labelData, signed char desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<unsigned char>(unsigned char* labelData, unsigned char desiredValue, short* agreement, int size, hipStream_t* stream);



template< class T >
void CUDA_IncrementInformation(T* labelData, T desiredValue, short* agreement, int size, hipStream_t* stream){
    T* GPUBuffer = 0;

	hipMalloc((void**) &GPUBuffer, sizeof(T)*size);
	hipMemcpyAsync( GPUBuffer, labelData, sizeof(T)*size, hipMemcpyHostToDevice, *stream );

	#ifdef DEBUG_VTKCUDA_IALP
		hipDeviceSynchronize();
		printf( "CUDA_IncrementInformation: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif

	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	kern_IncrementBuffer<T><<<grid,threads,0,*stream>>>(GPUBuffer, desiredValue, agreement, size);
	hipFree(GPUBuffer);

	#ifdef DEBUG_VTKCUDA_IALP
		hipDeviceSynchronize();
		printf( "CUDA_IncrementInformation: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

__global__ void kern_ZeroOutBuffer(short* buffer, int size){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if( idx < size ) buffer[idx] = 0;
}

void CUDA_GetRelevantBuffers(short** agreement, float** output, int size, hipStream_t* stream){
	hipMalloc((void**) agreement, sizeof(short)*size);
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	kern_ZeroOutBuffer<<<grid,threads,0,*stream>>>(*agreement,size);
	hipMalloc((void**) output, sizeof(float)*size);

	#ifdef DEBUG_VTKCUDA_IALP
		hipDeviceSynchronize();
		printf( "CUDA_GetRelevantBuffers: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

void CUDA_CopyBackResult(float* GPUBuffer, float* CPUBuffer, int size, hipStream_t* stream){
	hipDeviceSynchronize();
	hipMemcpy( CPUBuffer, GPUBuffer, sizeof(float)*size, hipMemcpyDeviceToHost );
	hipFree(GPUBuffer);

	#ifdef DEBUG_VTKCUDA_IALP
		hipDeviceSynchronize();
		printf( "CUDA_CopyBackResult: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

__global__ void kern_LogBuffer(short* agreement, float* output, float maxOut, int size, short max){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	float locAgreement = (float) agreement[idx];
	float logValue = (locAgreement > 0.0f) ? log((float)max)-log(locAgreement): maxOut;
	logValue = (logValue > 0.0f) ? logValue : 0.0f;
	logValue = (logValue < maxOut) ? logValue: maxOut;
	if( idx < size ) output[idx] = logValue;
}

__global__ void kern_NormLogBuffer(short* agreement, float* output, float maxOut, int size, short max){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	float locAgreement = (float) agreement[idx];
	float logValue = (locAgreement > 0.0f) ? log((float)max)-log(locAgreement): maxOut;
	logValue = (logValue > 0.0f) ? logValue : 0.0f;
	logValue = (logValue < maxOut) ? logValue / maxOut: 1.0f;
	if( idx < size ) output[idx] = logValue;
}

__global__ void kern_ProbBuffer(short* agreement, float* output, int size, short max){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	short locAgreement = agreement[idx];
	float probValue = (float) locAgreement / (float) max;
	probValue = (probValue < 1.0f) ? probValue: 1.0f;
	if( idx < size ) output[idx] = probValue;
}

void CUDA_ConvertInformation(short* agreement, float* output, float maxOut, int size, short max, short flags, hipStream_t* stream){
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	if( flags & 1 )
		if( flags & 2)
			kern_NormLogBuffer<<<grid,threads,0,*stream>>>(agreement, output, maxOut, size, max);
		else
			kern_LogBuffer<<<grid,threads,0,*stream>>>(agreement, output, maxOut, size, max);
	else
		kern_ProbBuffer<<<grid,threads,0,*stream>>>(agreement, output, size, max);

	#ifdef DEBUG_VTKCUDA_IALP
		hipDeviceSynchronize();
		printf( "CUDA_ConvertInformation: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}
