#include "hip/hip_runtime.h"
#include "CUDA_atlasprobability.h"
#include "CUDA_commonKernels.h"
#include "stdio.h"
#include "hip/hip_runtime.h"

//#define DEBUG_VTKCUDA_IALP

template void CUDA_IncrementInformation<float>(float* labelData, float desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<double>(double* labelData, double desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<long>(long* labelData, long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<unsigned long>(unsigned long* labelData, unsigned long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<long long>(long long* labelData, long long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<unsigned long long>(unsigned long long* labelData, unsigned long long desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<int>(int* labelData, int desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<unsigned int>(unsigned int* labelData, unsigned int desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<short>(short* labelData, short desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<unsigned short>(unsigned short* labelData, unsigned short desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<char>(char* labelData, char desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<signed char>(signed char* labelData, signed char desiredValue, short* agreement, int size, hipStream_t* stream);
template void CUDA_IncrementInformation<unsigned char>(unsigned char* labelData, unsigned char desiredValue, short* agreement, int size, hipStream_t* stream);

template< class T >
void CUDA_IncrementInformation(T* labelData, T desiredValue, short* agreement, int size, hipStream_t* stream){
    T* GPUBuffer = 0;

	hipMalloc((void**) &GPUBuffer, sizeof(T)*size);
	hipMemcpyAsync( GPUBuffer, labelData, sizeof(T)*size, hipMemcpyHostToDevice, *stream );

	#ifdef DEBUG_VTKCUDA_IALP
		hipDeviceSynchronize();
		printf( "CUDA_IncrementInformation: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif

	dim3 threads(NUMTHREADS,1,1);
	dim3 grid = GetGrid(size);
	IncrementBuffer<T><<<grid,threads,0,*stream>>>(GPUBuffer, desiredValue, agreement, size);
	hipFree(GPUBuffer);

	#ifdef DEBUG_VTKCUDA_IALP
		hipDeviceSynchronize();
		printf( "CUDA_IncrementInformation: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

void CUDA_GetRelevantBuffers(short** agreement, float** output, int size, hipStream_t* stream){
	hipMalloc((void**) agreement, sizeof(short)*size);
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid( (size-1)/NUMTHREADS + 1, 1, 1);
	ZeroOutBuffer<<<grid,threads,0,*stream>>>(*agreement,size);
	hipMalloc((void**) output, sizeof(float)*size);

	#ifdef DEBUG_VTKCUDA_IALP
		hipDeviceSynchronize();
		printf( "CUDA_GetRelevantBuffers: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

void CUDA_CopyBackResult(float* GPUBuffer, float* CPUBuffer, int size, hipStream_t* stream){
	hipDeviceSynchronize();
	hipMemcpy( CPUBuffer, GPUBuffer, sizeof(float)*size, hipMemcpyDeviceToHost );
	hipFree(GPUBuffer);

	#ifdef DEBUG_VTKCUDA_IALP
		hipDeviceSynchronize();
		printf( "CUDA_CopyBackResult: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}

__global__ void kern_ConvertBuffer(short* agreement, float* output, int size ){
	int idx = CUDASTDOFFSET;
	float locAgreement = (float) agreement[idx];
	if( idx < size ) output[idx] = locAgreement;
}

__global__ void kern_LogBuffer(float* agreement, float* output, float maxOut, int size, short max){
	int idx = CUDASTDOFFSET;
	float locAgreement = (float) agreement[idx];
	float logValue = (locAgreement > 0.0f) ? log((float)max)-log(locAgreement): maxOut;
	logValue = (logValue > 0.0f) ? logValue : 0.0f;
	logValue = (logValue < maxOut) ? logValue: maxOut;
	if( idx < size ) output[idx] = logValue;
}

__global__ void kern_NormLogBuffer(float* agreement, float* output, float maxOut, int size, short max){
	int idx = CUDASTDOFFSET;
	float locAgreement = (float) agreement[idx];
	float logValue = (locAgreement > 0.0f) ? log((float)max)-log(locAgreement): maxOut;
	logValue = (logValue > 0.0f) ? logValue : 0.0f;
	logValue = (logValue < maxOut) ? logValue / maxOut: 1.0f;
	if( idx < size ) output[idx] = logValue;
}

__global__ void kern_ProbBuffer(float* agreement, float* output, int size, short max){
	int idx = CUDASTDOFFSET;
	float locAgreement = agreement[idx];
	float probValue = (float) locAgreement / (float) max;
	probValue = (probValue < 1.0f) ? probValue: 1.0f;
	if( idx < size ) output[idx] = probValue;
}

__global__ void kern_BlurBuffer(float* agreement, float* output, int size, int spread, int dim){
	int idx = CUDASTDOFFSET;
	int x = (idx / spread) % dim;
	float curr = 0.7865707f * agreement[idx];
	float down = 0.1064508f * agreement[idx-spread];
	float up   = 0.1064508f * agreement[idx+spread];
	float newVal = curr + (x > 0 ? down : 0.0f) + (x < dim-1 ? up : 0.0f);
	if( idx < size ) output[idx] = newVal;
}

void CUDA_ConvertInformation(short* agreement, float* output, float maxOut, int size, short max, short flags, int gaussWidth[], int imageDims[], hipStream_t* stream){
	dim3 threads(NUMTHREADS,1,1);
	dim3 grid = GetGrid(size);
	
	//Gaussian smooth results
	float* floatAgreement = 0;
	hipMalloc( &floatAgreement, sizeof(float)*NUMTHREADS );
	kern_ConvertBuffer<<<grid,threads,0,*stream>>>(agreement, floatAgreement, size);
	if( flags & 3 ){
		while( gaussWidth[0] > 0 && gaussWidth[1] > 0 && gaussWidth[2] > 0 ){
			if( gaussWidth[0] > 0 ){
				kern_BlurBuffer<<<grid,threads,0,*stream>>>(floatAgreement, floatAgreement, size, 1, imageDims[0] );
				gaussWidth[0]--;
			}
			if( gaussWidth[1] > 0 ){
				kern_BlurBuffer<<<grid,threads,0,*stream>>>(floatAgreement, floatAgreement, size, imageDims[0], imageDims[1] );
				gaussWidth[1]--;
			}
			if( gaussWidth[2] > 0 ){
				kern_BlurBuffer<<<grid,threads,0,*stream>>>(floatAgreement, floatAgreement, size, imageDims[0]*imageDims[1], imageDims[2] );
				gaussWidth[2]--;
			}
		}
	}

	if( flags & 1 )
		if( flags & 2)
			kern_NormLogBuffer<<<grid,threads,0,*stream>>>(floatAgreement, output, maxOut, size, max);
		else
			kern_LogBuffer<<<grid,threads,0,*stream>>>(floatAgreement, output, maxOut, size, max);
	else
		kern_ProbBuffer<<<grid,threads,0,*stream>>>(floatAgreement, output, size, max);

	hipFree(agreement);
	hipFree(floatAgreement);

	#ifdef DEBUG_VTKCUDA_IALP
		hipDeviceSynchronize();
		printf( "CUDA_ConvertInformation: " );
		printf( hipGetErrorString( hipGetLastError() ) );
		printf( "\n" );
	#endif
}
