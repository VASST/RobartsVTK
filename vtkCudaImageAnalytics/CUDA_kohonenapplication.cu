#include "hip/hip_runtime.h"
#include "CUDA_kohonenapplication.h"
#include <float.h>
#include <stdio.h>

__constant__ Kohonen_Application_Information info;
texture<float, 3, hipReadModeElementType> Kohonen_Map;

#define NUM_THREADS 256


__global__ void ProcessImageToMapFirst(float* KohonenMap, float* InputData, float2* OutputData, float* OutputWeight, float2* OutputNearest, float* OutputDistance ){

	float currWeight = 0.0f;
	float2 weightedIndex = {0.0f, 0.0f};
	float2 minIndex = {0.0f, 0.0f};
	float minDistance = FLT_MAX;

	//save off the index of the closest map point
	int kOffset = threadIdx.x + blockDim.x * blockIdx.x;
	int bufferSize = info.VolumeSize[0]*info.VolumeSize[1]*info.VolumeSize[2];
	if( kOffset < bufferSize ) OutputData[kOffset] = weightedIndex;
	if( kOffset < bufferSize ) OutputWeight[kOffset] = currWeight;
	if( kOffset < bufferSize ) OutputNearest[kOffset] = minIndex;
	if( kOffset < bufferSize ) OutputDistance[kOffset] = minDistance;

}

__global__ void ProcessImageToMapRepeat(float* KohonenMap, float* InputData, float2* OutputData, float* OutputWeight, float2* OutputNearest, float* OutputDistance, int KohonenRow){

	__shared__ float MapPoint[NUM_THREADS];

	int kOffset = threadIdx.x + blockDim.x * blockIdx.x;

	float currWeight = OutputWeight[kOffset];
	float2 weightedIndex = OutputData[kOffset];
	float2 minIndex = OutputNearest[kOffset];
	float minDistance = OutputDistance[kOffset];
	
	int bufferSize = info.VolumeSize[0]*info.VolumeSize[1]*info.VolumeSize[2];
	int count = 0;
	for(int i = 0; i < info.KohonenMapSize[0]; i++){
		
		//load data into shared memory
		__syncthreads();
		if(count == 0){
			MapPoint[threadIdx.x] = KohonenMap[i*info.NumberOfDimensions+threadIdx.x];
		}
		__syncthreads();

		//find the distance to the map point
		float currDistance = 0.0f;
		for(int j = 0; j < info.NumberOfDimensions; j++){
			//float value = info.Weights[j]*(InputData[j*bufferSize+kOffset] - MapPoint[j]);
			float value = info.Weights[j]*(InputData[j*bufferSize+kOffset] - MapPoint[count*info.NumberOfDimensions+j]);
			//float value = info.Weights[j]*(InputData[j*bufferSize+kOffset] - KohonenMap[i*info.NumberOfDimensions+j]);
			currDistance += value*value;
		}

		//tell if we need to update the shared memory buffer
		count++;
		if( (count+1)*info.NumberOfDimensions > NUM_THREADS-1)
			count = 0;

		//if less than the minimum, save it
		weightedIndex.x += exp( -1.0f * currDistance ) * (float) i;
		weightedIndex.y += exp( -1.0f * currDistance ) * (float) KohonenRow;
		currWeight += exp( -1.0f * currDistance );
		minIndex.x = (minDistance <= currDistance) ? minIndex.x : (float) i;
		minIndex.y = (minDistance <= currDistance) ? minIndex.y : (float) KohonenRow;
		minDistance = (minDistance <= currDistance) ? minDistance : currDistance;

	}

	//save off the index of the closest map point
	if( kOffset < bufferSize ) OutputData[kOffset] = weightedIndex;
	if( kOffset < bufferSize ) OutputWeight[kOffset] = currWeight;
	if( kOffset < bufferSize ) OutputNearest[kOffset] = minIndex;
	if( kOffset < bufferSize ) OutputDistance[kOffset] = minDistance;

}

__global__ void NormalizeImage( float2* OutputData, float* OutputWeight, float2* OutputNearest, float* OutputDistance ){
	int kOffset = threadIdx.x + blockDim.x * blockIdx.x;
	float currWeight = OutputWeight[kOffset];
	float2 appIndex = OutputData[kOffset];
	float2 minIndex = OutputNearest[kOffset];
	appIndex.x = (currWeight > (appIndex.x / (float) info.KohonenMapSize[0]) ) ? appIndex.x / currWeight : minIndex.x;
	appIndex.y = (currWeight > (appIndex.y / (float) info.KohonenMapSize[1]) ) ? appIndex.y / currWeight : minIndex.y;
	int bufferSize = info.VolumeSize[0]*info.VolumeSize[1]*info.VolumeSize[2];
	if( kOffset < bufferSize ) OutputData[kOffset] = appIndex;
}

void CUDAalgo_applyKohonenMap( float* inputData, float* inputKohonen, float* outputData,
								Kohonen_Application_Information& information,
								hipStream_t* stream ){

	//copy information to GPU
	hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &information, sizeof(Kohonen_Application_Information) );

	//translate data onto device
	float* device_KohonenMap = 0;
	hipMalloc( (void**) &device_KohonenMap, sizeof(float)*information.KohonenMapSize[0]*information.KohonenMapSize[1]*information.NumberOfDimensions );
	hipMemcpyAsync( device_KohonenMap, inputKohonen, sizeof(float)*information.KohonenMapSize[0]*information.KohonenMapSize[1]*information.NumberOfDimensions, hipMemcpyHostToDevice, *stream );

	//rearrange image data to be easier to work with (should parallelize)
	float* device_InputData = 0;
	hipMalloc( (void**) &device_InputData, sizeof(float)*information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2]*information.NumberOfDimensions );
	float* inputTransposed = new float[information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2]*information.NumberOfDimensions];
	for( int i = 0; i < information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2]; i++ ){
		for( int j = 0; j < information.NumberOfDimensions; j++ ){
			int inIndex = i * information.NumberOfDimensions + j;
			int outIndex = j * information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2] + i;
			inputTransposed[outIndex] = inputData[inIndex];
		}
	}
	hipMemcpyAsync( device_InputData, inputTransposed, sizeof(float)*information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2]*information.NumberOfDimensions, hipMemcpyHostToDevice, *stream );
	delete[] inputTransposed;
	
	float2* device_OutputData = 0;
	hipMalloc( (void**) &device_OutputData, sizeof(float2)*information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2] );
	float* device_OutputWeight = 0;
	hipMalloc( (void**) &device_OutputWeight, sizeof(float)*information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2] );
	float2* device_OutputNearest = 0;
	hipMalloc( (void**) &device_OutputNearest, sizeof(float2)*information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2] );
	float* device_OutputDistance = 0;
	hipMalloc( (void**) &device_OutputDistance, sizeof(float)*information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2] );

	//apply the map
	dim3 grid((information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2] + NUM_THREADS - 1) / NUM_THREADS,1,1);
	dim3 threads(NUM_THREADS,1,1);
	ProcessImageToMapFirst<<<grid, threads, 0, *stream >>>(device_KohonenMap, device_InputData, device_OutputData, device_OutputWeight, device_OutputNearest, device_OutputDistance );
	hipStreamSynchronize(*stream);
	for( int i = 0; i < information.KohonenMapSize[1]; i++ ){
		ProcessImageToMapRepeat<<<grid, threads, 0, *stream >>>(device_KohonenMap + i*information.KohonenMapSize[0]*information.NumberOfDimensions,
									device_InputData, device_OutputData, device_OutputWeight, device_OutputNearest, device_OutputDistance, i );
		hipStreamSynchronize(*stream);
	}
	NormalizeImage<<<grid, threads, 0, *stream >>>(device_OutputData, device_OutputWeight, device_OutputNearest, device_OutputDistance );
		
	printf( "Apply Map: " );
	hipStreamSynchronize(*stream);
	printf( hipGetErrorString( hipGetLastError() ) );
	printf( "\n" );

	//copy results back
	hipMemcpyAsync( outputData, device_OutputData, sizeof(float2)*information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2], hipMemcpyDeviceToHost, *stream );
	hipStreamSynchronize(*stream);

	//remove allocated memory
	hipFree(device_KohonenMap);
	hipFree(device_InputData);
	hipFree(device_OutputData);
	hipFree(device_OutputWeight);
	hipFree(device_OutputNearest);
	hipFree(device_OutputDistance);
}