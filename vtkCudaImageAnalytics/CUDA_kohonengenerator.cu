#include "CUDA_kohonengenerator.h"

//3D input data (read-only texture with corresponding opague device memory back)
texture<float, 3, hipReadModeElementType> kohonen_input_texture;
hipArray* kohonen_input_array = 0;

//parameters held in constant memory
__constant__ Kohonen_Generator_Information info;

void CUDAsetup_loadNDImage( hipStream_t* stream ){

}

void CUDAsetup_loadNDImage( float* image, Kohonen_Generator_Information& information, hipStream_t* stream){

}

void CUDAalgo_generateKohonenMap( float* outputKohonen, Kohonen_Generator_Information& information, hipStream_t* stream ){

}
