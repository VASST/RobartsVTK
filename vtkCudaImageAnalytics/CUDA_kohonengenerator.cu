#include "hip/hip_runtime.h"
#include "CUDA_kohonengenerator.h"
#include "CUDA_commonKernels.h"
#include <float.h>
#include <stdio.h>
#include <time.h>

#define DEBUGGING

#define VTK_KSOMGen_EPSILON 0.0000001f

//parameters held in constant memory
__constant__ Kohonen_Generator_Information info;
__constant__ float SamplePoint[MAX_DIMENSIONALITY];

__global__ void ProcessSample(float* KohonenMap, float* DistanceBuffer, short2* IndexBuffer, float* WeightBuffer, int mapSizeX, int mapSizeY ){

	__shared__ float SamplePointLocal[MAX_DIMENSIONALITY];

	//get sample co-ordinates in buffer
	int kOffset = CUDASTDOFFSET;
	if(threadIdx.x < MAX_DIMENSIONALITY){
		SamplePointLocal[threadIdx.x] = SamplePoint[threadIdx.x];
	}
	__syncthreads();
	
	//calculate the distance
	float distance = 0.0f;
	float penalty = KohonenMap[kOffset] + VTK_KSOMGen_EPSILON;
	penalty *= penalty;
	int bufferSize = mapSizeX * mapSizeY;
	for(int i = 0; i < info.NumberOfDimensions; i++){
		float weight = KohonenMap[(2*i+2)*bufferSize+kOffset];
		float value = (KohonenMap[(2*i+1)*bufferSize+kOffset] - SamplePointLocal[i]);
		distance += value*value / weight;
		penalty *= weight;
	}
	distance += 0.5f * log(penalty);
	__syncthreads();

	if( kOffset < bufferSize ) DistanceBuffer[kOffset] = distance;
	if( kOffset < bufferSize ) WeightBuffer[kOffset] = exp(-distance);
	short2 index = {kOffset % mapSizeX, kOffset / mapSizeX };
	if( kOffset < bufferSize ) IndexBuffer[kOffset] = index;

}

__global__ void DoubleMapSizeInX( float* KohonenMap, float* tempStore, int currMapSizeX, int currMapSizeY ){

	int kOffset = CUDASTDOFFSET;

	//double size in X direction
	int bufferSize = currMapSizeX * currMapSizeY;
	int xIndex = kOffset % currMapSizeX;
	for(int i = 0; i < 2*info.NumberOfDimensions+1; i++){
		float valueOld = KohonenMap[i*bufferSize+kOffset];
		float valueNeighbour = KohonenMap[i*bufferSize+kOffset+1];
		float difference = (xIndex != currMapSizeX-1) ? valueNeighbour - valueOld : 0.0f;
		
		if(i){
			float2 outputValue = {valueOld, valueOld + 0.5f * difference};
			if( kOffset < bufferSize )((float2*) tempStore)[i*bufferSize+kOffset] = outputValue;
		}else{
			float2 outputValue = {valueOld*0.5f, valueOld*0.5f};
			if( kOffset < bufferSize )((float2*) tempStore)[i*bufferSize+kOffset] = outputValue;
		}
	}
}

__global__ void DoubleMapSizeInY( float* KohonenMap, float* tempStore, int currMapSizeX, int currMapSizeY ){
	int kOffset = CUDASTDOFFSET;
	int bufferSize = currMapSizeX * currMapSizeY;
	
	//double size in Y direction
	int xIndex = kOffset % currMapSizeX;
	int yIndex = kOffset / currMapSizeX;
	for(int i = 0; i < 2*info.NumberOfDimensions+1; i++){
		float valueOld = tempStore[i*bufferSize+kOffset];
		float valueNeighbour = tempStore[i*bufferSize+kOffset+currMapSizeX];
		float difference = (yIndex != currMapSizeY-1) ? valueNeighbour - valueOld : 0.0f;

		if(i){
			if( kOffset < bufferSize ) KohonenMap[i*2*bufferSize+xIndex+currMapSizeX*2*yIndex] = valueOld;
			if( kOffset < bufferSize ) KohonenMap[i*2*bufferSize+xIndex+currMapSizeX*(2*yIndex+1)] = valueOld + 0.5f * difference;
		}else{
			if( kOffset < bufferSize ) KohonenMap[xIndex+currMapSizeX*2*yIndex] = valueOld * 0.5f;
			if( kOffset < bufferSize ) KohonenMap[xIndex+currMapSizeX*(2*yIndex+1)] = valueOld *0.5f;

		}
	}

}

template <unsigned int blockSize>
__global__ void reduce6(float *g_idata, float *g_odata, short2 * i_idata, short2 *i_odata, unsigned int n)
{
	__shared__ float sdata[blockSize];
	__shared__ short2 sindex[blockSize];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;
	sdata[tid] = FLT_MAX;
	sindex[tid].x = 0;
	sindex[tid].y = 0;
	
	while (i < n) {
		if( sdata[tid] >= g_idata[i] ){
			sdata[tid] = g_idata[i];
			sindex[tid] = i_idata[i];
		}
		if( sdata[tid] >= g_idata[i+blockSize] ){
			sdata[tid] = g_idata[i+blockSize];
			sindex[tid] = i_idata[i+blockSize];
		}
		i += gridSize;
		__syncthreads();
	}
	
	if (blockSize >= 512) { if (tid < 256) {
			if( sdata[tid] >= sdata[tid + 256] ){
				sdata[tid] = sdata[tid + 256];
				sindex[tid] = sindex[tid + 256];
			}
	} __syncthreads(); }

	if (blockSize >= 256) { if (tid < 128) {
			if( sdata[tid] >= sdata[tid + 128] ){
				sdata[tid] = sdata[tid + 128];
				sindex[tid] = sindex[tid + 128];
			}
	} __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) {
			if( sdata[tid] >= sdata[tid + 64] ){
				sdata[tid] = sdata[tid + 64];
				sindex[tid] = sindex[tid + 64];
			}
	} __syncthreads(); }
	
	if (tid < 32) {
		if (blockSize >= 64){
			if( sdata[tid] >= sdata[tid + 32] ){
				sdata[tid] = sdata[tid + 32];
				sindex[tid] = sindex[tid + 32];
			}
			__syncthreads();
		}
		if (blockSize >= 32){
			if( sdata[tid] >= sdata[tid + 16] ){
				sdata[tid] = sdata[tid + 16];
				sindex[tid] = sindex[tid + 16];
			}
			__syncthreads();
		}
		if (blockSize >= 16){
			if( sdata[tid] >= sdata[tid + 8] ){
				sdata[tid] = sdata[tid + 8];
				sindex[tid] = sindex[tid + 8];
			}
			__syncthreads();
		}
		if (blockSize >=  8){
			if( sdata[tid] >= sdata[tid + 4] ){
				sdata[tid] = sdata[tid + 4];
				sindex[tid] = sindex[tid + 4];
			}
			__syncthreads();
		}
		if (blockSize >=  4){
			if( sdata[tid] >= sdata[tid + 2] ){
				sdata[tid] = sdata[tid + 2];
				sindex[tid] = sindex[tid + 2];
			}
			__syncthreads();
		}
		if (blockSize >=  2){
			if( sdata[tid] >= sdata[tid + 1] ){
				sdata[tid] = sdata[tid + 1];
				sindex[tid] = sindex[tid + 1];
			}
			__syncthreads();
		}
	}
	if (tid == 0){
		g_odata[0] = sdata[0];
		i_odata[0] = sindex[0];
	}
}

__global__ void FindMinSample( float* DistanceBuffer, short2* IndexBuffer, int spread, int mapSizeX, int mapSizeY ){
	
	int kOffset = CUDASTDOFFSET;
	float distance1 = DistanceBuffer[kOffset];
	float distance2 = DistanceBuffer[kOffset+spread];
	short2 index1 = IndexBuffer[kOffset];
	short2 index2 = IndexBuffer[kOffset+spread];

	if( kOffset+spread < mapSizeX * mapSizeY ){
		DistanceBuffer[kOffset] = (distance1 < distance2) ? distance1 : distance2;
		IndexBuffer[kOffset] = (distance1 < distance2) ? index1 : index2;
	}

}

__global__ void UpdateWeights( float* KohonenMap, short2 minIndex, float weightTot, float mAlpha, float mNeigh, float vAlpha, float vNeigh, float wAlpha, int mapSizeX, int mapSizeY ){

	__shared__ float SamplePointLocal[MAX_DIMENSIONALITY];

	//get sample co-ordinates in buffer
	int kOffset = CUDASTDOFFSET;
	short2 currIndex = {kOffset % mapSizeX, kOffset / mapSizeX };
	if(threadIdx.x < MAX_DIMENSIONALITY)
		SamplePointLocal[threadIdx.x] = SamplePoint[threadIdx.x];
	
	//figure out the multipliers
	float mMultiplier = mAlpha * exp( -((currIndex.x-minIndex.x)*(currIndex.x-minIndex.x) + (currIndex.y-minIndex.y)*(currIndex.y-minIndex.y) ) / mNeigh );
	float vMultiplier = vAlpha * exp( -((currIndex.x-minIndex.x)*(currIndex.x-minIndex.x) + (currIndex.y-minIndex.y)*(currIndex.y-minIndex.y) ) / vNeigh );
	float wMultiplier = wAlpha;

	//adjust the weights
	float distance = 0.0f;
	float weight = KohonenMap[kOffset];
	float penalty = weight + VTK_KSOMGen_EPSILON;
	penalty *= penalty;
	int bufferSize = mapSizeX * mapSizeY;
	for(int i = 0; i < info.NumberOfDimensions; i++){
		float mean = KohonenMap[(2*i+1)*bufferSize+kOffset];
		float variance = KohonenMap[(2*i+2)*bufferSize+kOffset];
		
		distance += (SamplePointLocal[i]-mean)*(SamplePointLocal[i]-mean) / variance;
		penalty *= variance;

		float newMean = (1.0f-mMultiplier)*mean + mMultiplier*SamplePointLocal[i];
		float newVariance = (1.0f-vMultiplier)*variance + vMultiplier*(SamplePointLocal[i]-mean)*(SamplePointLocal[i]-mean);

		//float value = SamplePointLocal[i];
		if( kOffset < bufferSize ) KohonenMap[(2*i+1)*bufferSize+kOffset] = newMean;
		if( kOffset < bufferSize ) KohonenMap[(2*i+2)*bufferSize+kOffset] = newVariance;
		__syncthreads();
	}
	distance += 0.5f * log(penalty);
	float newWeight = (exp(-distance) + VTK_KSOMGen_EPSILON) / weightTot;
	if( kOffset < bufferSize ) KohonenMap[kOffset] = weight + wMultiplier*(newWeight-weight);

}

void getMinimum(int size, int threads, int blocks, float *d_idata, float *d_odata, short2* d_iindex, short2* d_oindex, hipStream_t* stream ){

    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    int smemSize = (threads <= 32) ? 2 * threads * (sizeof(float)+sizeof(short2)) : threads * (sizeof(float)+sizeof(short2));
	switch (threads)
	{
	case 512:
		reduce6<512><<< dimGrid, dimBlock, smemSize, *stream >>>(d_idata, d_odata, d_iindex, d_oindex, size); break;
	case 256:
		reduce6<256><<< dimGrid, dimBlock, smemSize, *stream >>>(d_idata, d_odata, d_iindex, d_oindex, size); break;
	case 128:
		reduce6<128><<< dimGrid, dimBlock, smemSize, *stream >>>(d_idata, d_odata, d_iindex, d_oindex, size); break;
	case 64:
		reduce6< 64><<< dimGrid, dimBlock, smemSize, *stream >>>(d_idata, d_odata, d_iindex, d_oindex, size); break;
	case 32:
		reduce6< 32><<< dimGrid, dimBlock, smemSize, *stream >>>(d_idata, d_odata, d_iindex, d_oindex, size); break;
	case 16:
		reduce6< 16><<< dimGrid, dimBlock, smemSize, *stream >>>(d_idata, d_odata, d_iindex, d_oindex, size); break;
	case 8:
		reduce6< 8><<< dimGrid, dimBlock, smemSize, *stream >>>(d_idata, d_odata, d_iindex, d_oindex, size); break;
	case 4:
		reduce6< 4><<< dimGrid, dimBlock, smemSize, *stream >>>(d_idata, d_odata, d_iindex, d_oindex, size); break;
	case 2:
		reduce6< 2><<< dimGrid, dimBlock, smemSize, *stream >>>(d_idata, d_odata, d_iindex, d_oindex, size); break;
	case 1:
		reduce6< 1><<< dimGrid, dimBlock, smemSize, *stream >>>(d_idata, d_odata, d_iindex, d_oindex, size); break;
	}

}

unsigned int gcd(unsigned int r, unsigned int n){
	if( r < n ) return gcd( n, r );
	if( n == 0 ) return r;
	return gcd( n, r % n );
}

unsigned int random_prime(unsigned int n){
     unsigned int r = rand();
	 unsigned int t;
     while ((t = gcd(r, n)) > 1)
         r /= t;
     return r;
}

void CUDAalgo_KSOMInitialize( double* range, Kohonen_Generator_Information& information, int* currentMapSize,
								float** device_KohonenMap, float** device_tempSpace,
								float** device_DistanceBuffer, short2** device_IndexBuffer, float** device_WeightBuffer,
								float meansWidth, float varsWidth, hipStream_t* stream ){
	
	//make sure parametes are in reasonable range
	meansWidth = max( meansWidth, FLT_MIN );
	varsWidth = max( varsWidth, FLT_MIN );

	//copy information to GPU
	hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &information, sizeof(Kohonen_Generator_Information) );

	//find minimum starting size
	float neighbourhood = min( meansWidth, varsWidth );
	currentMapSize[0] = currentMapSize[1] = 2;
	currentMapSize[2] = 1;
	while( neighbourhood * (double) currentMapSize[0] <= 8.0 && currentMapSize[0] < information.KohonenMapSize[0] ) currentMapSize[0] += currentMapSize[0];
	if( currentMapSize[0] > information.KohonenMapSize[0] ) currentMapSize[0] = information.KohonenMapSize[0];
	while( neighbourhood * (double) currentMapSize[1] <= 8.0 && currentMapSize[1] < information.KohonenMapSize[1] ) currentMapSize[1] += currentMapSize[1];
	if( currentMapSize[1] > information.KohonenMapSize[1] ) currentMapSize[1] = information.KohonenMapSize[1];
	int MapSize = information.KohonenMapSize[0]*information.KohonenMapSize[1];
	int currMapSize = currentMapSize[0]*currentMapSize[1];
	#ifdef DEBUGGING
		printf("Updating size to (%d,%d)\n", currentMapSize[0],currentMapSize[1]);
	#endif

	//allocate a distance buffer
	hipMalloc( (void**) device_DistanceBuffer, sizeof(float)*MapSize );
	hipMalloc( (void**) device_IndexBuffer, sizeof(short2)*MapSize );
	hipMalloc( (void**) device_WeightBuffer, sizeof(float)*MapSize );
	
	//create buffer for the Kohonen map
	hipMalloc( (void**) device_KohonenMap, sizeof(float)*MapSize*(2*information.NumberOfDimensions+1) );
	hipMalloc( (void**) device_tempSpace, sizeof(float)*MapSize*(2*information.NumberOfDimensions+1) );

	//initialize weights
	dim3 grid((currentMapSize[0]*currentMapSize[1]-1)/NUMTHREADS+1, 1, 1);
	dim3 threads(NUMTHREADS, 1, 1);
	SetBufferToConst<<<grid, threads, 0, *stream>>>(*device_KohonenMap, (float) (1.0 / (double) currMapSize), currMapSize);
	for(int j = 0; j < information.NumberOfDimensions; j++ ){
		SetBufferToRandom<<<grid, threads, 0, *stream>>>((*device_KohonenMap)+(2*j+1)*currMapSize, (float) range[2*j+1], (float) range[2*j], currMapSize);
		SetBufferToConst<<<grid, threads, 0, *stream>>>((*device_KohonenMap)+(2*j+2)*currMapSize, information.Weights[j], currMapSize);
	}


}

void CUDAalgo_KSOMIteration( float** inputData,  char** maskData, int epoch,
								int* currentMapSize,
								float** device_KohonenMap, float** device_tempSpace,
								float** device_DistanceBuffer, short2** device_IndexBuffer, float** device_WeightBuffer,
								int* VolumeSize, int NumVolumes,
								Kohonen_Generator_Information& information,
								int BatchSize,
								float meansAlpha, float meansWidth,
								float varsAlpha, float varsWidth,
								float weiAlpha,
								hipStream_t* stream ){

	//make sure parameters are in a reasonable range
	meansAlpha = max( meansAlpha, FLT_MIN );
	meansWidth = max( meansWidth, FLT_MIN );
	varsAlpha = max( varsAlpha, FLT_MIN );
	varsWidth = max( varsWidth, FLT_MIN );
	weiAlpha = max( weiAlpha, FLT_MIN );

	dim3 grid((currentMapSize[0]*currentMapSize[1]-1)/NUMTHREADS+1, 1, 1);
	dim3 threads(NUMTHREADS, 1, 1);

	//make sure map is large enough
	float neighbourhood = min( meansWidth, varsWidth ) * (currentMapSize[0]+currentMapSize[1]) / 2;
	if( ((neighbourhood <= 8.0 ) && (currentMapSize[0] < information.KohonenMapSize[0])) ){
		grid = dim3 ((2*currentMapSize[0]*currentMapSize[1]-1)/NUMTHREADS+1, 1, 1);
		DoubleMapSizeInX<<<grid, threads, 0, *stream>>>( *device_KohonenMap, *device_tempSpace, currentMapSize[0], currentMapSize[1] );
		currentMapSize[0] *= 2;
		#ifdef DEBUGGING
			printf("Updating size to (%d,%d)\n", currentMapSize[0],currentMapSize[1]);
		#endif
	}
	if( ((neighbourhood <= 8.0) && (currentMapSize[1] < information.KohonenMapSize[1])) ){
		grid = dim3 ((2*currentMapSize[0]*currentMapSize[1]-1)/NUMTHREADS+1, 1, 1);
		DoubleMapSizeInY<<<grid, threads, 0, *stream>>>( *device_KohonenMap, *device_tempSpace, currentMapSize[0], currentMapSize[1] );
		currentMapSize[1] *= 2;
		#ifdef DEBUGGING
			printf("Updating size to (%d,%d)\n", currentMapSize[0],currentMapSize[1]);
		#endif
	}
	
	float meansNeigh = meansWidth * (currentMapSize[0]+currentMapSize[1]) / 2;
	float varsNeigh = varsWidth * (currentMapSize[0]+currentMapSize[1]) / 2;

	//update grid size
	grid = dim3((currentMapSize[0]*currentMapSize[1]-1)/NUMTHREADS+1, 1, 1);

	float* cpuWeights = new float[currentMapSize[0]*currentMapSize[1]];

	//train kohonen map
	if( BatchSize == -1 ){
	
		//generate a random iterator through [0,NumVolumes-1]
		int pictureIncrement = random_prime( NumVolumes ) % NumVolumes;
		int pictureInUse = rand() % NumVolumes;

		for( int picture = 0; picture < NumVolumes; picture++ ){

			//figure out what pseudo-random picture to grab
			pictureInUse = (pictureInUse + pictureIncrement) % NumVolumes;
			int NumVoxels = VolumeSize[3*pictureInUse]*VolumeSize[3*pictureInUse+1]*VolumeSize[3*pictureInUse+2];
				
			//generate a random iterator through [0,NumVolumes-1]
			int offsetIncrement = random_prime( NumVoxels ) % NumVoxels;
			int offsetInUse = rand() % NumVoxels;

			for( int sampleOffset = 0; sampleOffset < NumVoxels; sampleOffset++){
					
				//figure out what pseudo-random offset to grab
				offsetInUse = (offsetInUse + offsetIncrement) % NumVoxels;
				int sampleDimensionalOffset = information.NumberOfDimensions * offsetInUse;

				//if this is not a valid sample (ie: masked out) then try again
				if( maskData && (maskData[pictureInUse])[offsetInUse] == 0 )
					continue;

				//find the distance between each centroid and the sample
				hipMemcpyToSymbolAsync(HIP_SYMBOL(SamplePoint), &((inputData[pictureInUse])[sampleDimensionalOffset]),
										sizeof(float)*information.NumberOfDimensions );
				hipStreamSynchronize(*stream);
				ProcessSample<<<grid, threads, 0, *stream>>>(*device_KohonenMap, *device_DistanceBuffer, *device_IndexBuffer,
																*device_WeightBuffer, currentMapSize[0], currentMapSize[1]);

					
				//update the weights of each centroid
				short2 minIndex = {-1,-1};
				float distance = -1.0f;
				hipStreamSynchronize(*stream);
				hipMemcpy( &minIndex, *device_IndexBuffer, sizeof(short2), hipMemcpyDeviceToHost );
				hipMemcpy( &distance, *device_DistanceBuffer, sizeof(float), hipMemcpyDeviceToHost );

				//find the winning centroid
				for(int i = currentMapSize[0]*currentMapSize[1] / 2; i > NUMTHREADS; i = i/2){
					dim3 tempGrid( i>NUMTHREADS ? i/NUMTHREADS : 1, 1, 1);
					FindMinSample<<<tempGrid, threads, 0, *stream>>>(*device_DistanceBuffer, *device_IndexBuffer, i,
																		currentMapSize[0], currentMapSize[1]);
				}
				getMinimum( min(NUMTHREADS,currentMapSize[0]*currentMapSize[1]), min(NUMTHREADS,currentMapSize[0]*currentMapSize[1]), 1,
							*device_DistanceBuffer, *device_DistanceBuffer, *device_IndexBuffer, *device_IndexBuffer, stream );

				//update the weights of each centroid
				hipStreamSynchronize(*stream);
				hipMemcpy( &minIndex, *device_IndexBuffer, sizeof(short2), hipMemcpyDeviceToHost );
				long double weightTot = currentMapSize[0]*currentMapSize[1]*VTK_KSOMGen_EPSILON;
				hipMemcpy( cpuWeights, *device_WeightBuffer, sizeof(float)*currentMapSize[0]*currentMapSize[1], hipMemcpyDeviceToHost );
				for(int i = 0; i < currentMapSize[0]*currentMapSize[1]; i++)
					weightTot += (long double) cpuWeights[i];

				UpdateWeights<<<grid, threads, 0, *stream>>>(*device_KohonenMap, minIndex, (float) weightTot, meansAlpha, meansNeigh,
															 varsAlpha, varsNeigh, weiAlpha, currentMapSize[0], currentMapSize[1]);
			}
		}

		

	//if we are randomly sampling from the images
	}else{

		for( int batch = 0; batch < BatchSize; batch++ ){

			int sampleP = rand() % NumVolumes;
			int sampleX = rand() % VolumeSize[3*sampleP];
			int sampleY = rand() % VolumeSize[3*sampleP+1];
			int sampleZ = rand() % VolumeSize[3*sampleP+2];
			int sampleOffset = (sampleX + VolumeSize[3*sampleP] *( sampleY + VolumeSize[3*sampleP+1] * sampleZ ) );
			int sampleDimensionalOffset = information.NumberOfDimensions * sampleOffset;

			//if this is not a valid sample (ie: masked out) then try again
			if( maskData && (maskData[sampleP])[sampleOffset] == 0 ){
				batch--;
				continue;
			}

			//find the distance between each centroid and the sample
			hipMemcpyToSymbolAsync(HIP_SYMBOL(SamplePoint), &((inputData[sampleP])[sampleDimensionalOffset]), sizeof(float)*information.NumberOfDimensions );
			hipStreamSynchronize(*stream);
			ProcessSample<<<grid, threads, 0, *stream>>>(*device_KohonenMap, *device_DistanceBuffer, *device_IndexBuffer, *device_WeightBuffer, currentMapSize[0], currentMapSize[1]);
				
			//find the winning centroid
			for(int i = currentMapSize[0]*currentMapSize[1] / 2; i > NUMTHREADS; i = i/2){
				dim3 tempGrid( i>NUMTHREADS ? i/NUMTHREADS : 1, 1, 1);
				FindMinSample<<<tempGrid, threads, 0, *stream>>>(*device_DistanceBuffer, *device_IndexBuffer, i, currentMapSize[0], currentMapSize[1]);
			}
			getMinimum( min(NUMTHREADS,currentMapSize[0]*currentMapSize[1]), min(NUMTHREADS,currentMapSize[0]*currentMapSize[1]), 1,
						*device_DistanceBuffer, *device_DistanceBuffer, *device_IndexBuffer, *device_IndexBuffer, stream );

			//update the weights of each centroid
			short2 minIndex;
			hipMemcpyAsync( &minIndex, *device_IndexBuffer, sizeof(short2), hipMemcpyDeviceToHost, *stream );
			hipStreamSynchronize(*stream);
			hipMemcpy( cpuWeights, *device_WeightBuffer, sizeof(float)*currentMapSize[0]*currentMapSize[1], hipMemcpyDeviceToHost );
			long double weightTot = currentMapSize[0]*currentMapSize[1]*VTK_KSOMGen_EPSILON;
			for(int i = 0; i < currentMapSize[0]*currentMapSize[1]; i++)
				weightTot += (long double) cpuWeights[i];
			UpdateWeights<<<grid, threads, 0, *stream>>>(*device_KohonenMap, minIndex, (float) weightTot, meansAlpha, meansNeigh,
										varsAlpha, varsNeigh, weiAlpha, currentMapSize[0], currentMapSize[1]);
			
		}

	}
	
	#ifdef DEBUGGING
		printf("Finished epoch %d with:\n",epoch);
		printf("%d  M:(a,n) = (%f,%f,%f)\n",(int) time(NULL),meansAlpha, meansWidth, meansNeigh ); 
		printf("            V:(a,n) = (%f,%f,%f)\n",varsAlpha, varsWidth, varsNeigh);
		printf("            W:(a)   = (%f)\n",weiAlpha);
	#endif

	delete cpuWeights;
}

void CUDAalgo_KSOMOffLoad( float* outputKohonen, float** device_KohonenMap,
							float** device_tempSpace,
							float** device_DistanceBuffer, short2** device_IndexBuffer, float** device_WeightBuffer, 
							Kohonen_Generator_Information& information,
							hipStream_t* stream ){

	//remove distance buffer
	hipFree(*device_DistanceBuffer);
	hipFree(*device_IndexBuffer);
	hipFree(*device_WeightBuffer);

	//translate back data
	int MapSize = information.KohonenMapSize[0]*information.KohonenMapSize[1];
	float* tempKohonen = new float[MapSize*(2*information.NumberOfDimensions+1)];
	hipMemcpyAsync( tempKohonen, *device_KohonenMap, sizeof(float)*MapSize*(2*information.NumberOfDimensions+1),
					 hipMemcpyDeviceToHost, *stream );
	hipStreamSynchronize(*stream);
	hipFree(*device_KohonenMap);
	hipFree(*device_tempSpace);
	hipStreamSynchronize(*stream);

	int bufferJump = information.KohonenMapSize[0]*information.KohonenMapSize[1];
	for(int i = 0; i < information.KohonenMapSize[0]*information.KohonenMapSize[1]; i++)
		for( int j = 0; j < 2*information.NumberOfDimensions+1; j++ )
			outputKohonen[i*(2*information.NumberOfDimensions+1)+j] = tempKohonen[j*bufferJump+i];
	delete[] tempKohonen;

}