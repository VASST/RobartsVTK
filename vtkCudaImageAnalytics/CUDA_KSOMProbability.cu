#include "hip/hip_runtime.h"
#include "CUDA_KSOMProbability.h"
#include "CUDA_commonKernels.h"
#include <float.h>
#include <stdio.h>

__constant__ Kohonen_Probability_Information info;

__global__ void ProcessSample(float* InputData, float* KohonenMap, float* Buffer){

	__shared__ float SamplePointLocal[MAX_DIMENSIONALITY];

	//get sample co-ordinates in buffer
	int kOffset = CUDASTDOFFSET;
	if(threadIdx.x < MAX_DIMENSIONALITY){
		SamplePointLocal[threadIdx.x] = InputData[threadIdx.x];
	}
	__syncthreads();
	
	//calculate the distance
	float distance = 0.0f;
	float penalty = 1.0f;
	int bufferSize = info.KohonenMapSize[0] * info.KohonenMapSize[1];
	for(int i = 0; i < info.NumberOfDimensions; i++){
		float var = KohonenMap[(2*i+1)*bufferSize+kOffset];
		float value = (KohonenMap[(2*i)*bufferSize+kOffset] - SamplePointLocal[i]);
		distance += value * value * info.Scale / var;
		penalty *= var;
	}
	distance += 0.5 * log(penalty);

	//output weight
	//float weight = exp( -1.0f * distance );
	//if(kOffset < bufferSize) Buffer[kOffset] = weight;
	if(kOffset < bufferSize) Buffer[kOffset] = distance;
	
}

__global__ void SumOverLargeBufferLogBased( float* buffer, int spread, int size ){
	
	int offset = CUDASTDOFFSET;
	float value1 = buffer[offset];
	float value2 = buffer[offset+spread];
	
	float x = max(value1,value2);
	float n = min(value1,value2);
	float value = x - log( 1+exp(x-n) );

	if( offset+spread < size )
		buffer[offset] = value;

}

void CUDAalgo_applyProbabilityMaps( float* inputData, char* inputMask, float* inputKohonen, float** probabilityData,
									float** outputData, bool useProbData, bool useEntropy,
									Kohonen_Probability_Information& information, hipStream_t* stream ){

	//copy information to GPU
	hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &information, sizeof(Kohonen_Probability_Information) );

	//translate data onto device (need to transpose KSOM)
	int VolumeSize = information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2];
	int MapSize = information.KohonenMapSize[0]*information.KohonenMapSize[1];
	float* tempKohonen = new float[2*MapSize*information.NumberOfDimensions];
	int bufferJump = MapSize;
	for(int i = 0; i < MapSize; i++)
		for( int j = 0; j < 2*information.NumberOfDimensions; j++ )
			tempKohonen[j*bufferJump+i] = inputKohonen[i*2*information.NumberOfDimensions+j];
	float* device_KohonenMap = 0;
	hipMalloc( (void**) &device_KohonenMap, sizeof(float)*MapSize*2*information.NumberOfDimensions );
	hipMemcpy( device_KohonenMap, tempKohonen, sizeof(float)*MapSize*2*information.NumberOfDimensions, hipMemcpyHostToDevice );
	delete[] tempKohonen;

	//allocate a distance buffer
	float* device_BaseBuffer = 0;
	hipMalloc( (void**) &device_BaseBuffer, sizeof(float)*MapSize );
	float* device_WorkingBuffer = 0;
	hipMalloc( (void**) &device_WorkingBuffer, sizeof(float)*MapSize );

	//rearrange image data to be easier to work with (should parallelize)
	float* device_InputData = 0;
	hipMalloc( (void**) &device_InputData, sizeof(float)*VolumeSize*information.NumberOfDimensions );
	hipMemcpyAsync( device_InputData, inputData, sizeof(float)*VolumeSize*information.NumberOfDimensions, hipMemcpyHostToDevice, *stream );
	
	//copy probability buffers
	float* device_ProbabilityBuffer = 0;
	if( useProbData ){
		hipMalloc( (void**) &device_ProbabilityBuffer, sizeof(float)*MapSize*information.NumberOfLabels );
		for( int i = 0; i < information.NumberOfLabels; i++)
			hipMemcpyAsync( device_ProbabilityBuffer+i*MapSize, probabilityData[i], sizeof(float)*MapSize, hipMemcpyHostToDevice, *stream );
	}

	//apply the map
	dim3 grid = GetGrid(MapSize);
	dim3 threads(NUMTHREADS,1,1);
	for( int voxel = 0; voxel < VolumeSize; voxel++ ){
		
		//if we are not in the mask, ignore this voxel
		if( inputMask != 0 && inputMask[voxel] == 0 ){
			for( int i = 0; i < information.NumberOfLabels; i++)
				(outputData[i])[voxel] = FLT_MAX;
			continue;
		}

		//else, process it over the entire map
		int InputBufferOffset = voxel*information.NumberOfDimensions;
		ProcessSample<<<grid, threads, 0, *stream>>>(device_InputData+InputBufferOffset, device_KohonenMap, device_BaseBuffer );
		
		for( int i = 0; i < information.NumberOfLabels; i++){

			//multiply the basic amount with the probability buffer into the working buffer
			if( useProbData )
				MultiplyAndStoreBuffer<<<grid, threads, 0, *stream>>>(device_BaseBuffer, device_ProbabilityBuffer+i*MapSize, device_WorkingBuffer, MapSize );
			else
				CopyBuffers<<<grid, threads, 0, *stream>>>(device_WorkingBuffer, device_BaseBuffer, MapSize);

			//reduce working buffer by summation
			int j = 1;
			while( j < MapSize ) j += j;
			for(; j >= 1; j = j/2){
				dim3 tempGrid = GetGrid(j);
				SumOverLargeBufferLogBased<<<tempGrid, threads, 0, *stream>>>(device_WorkingBuffer,j,MapSize);
				hipStreamSynchronize(*stream);
			}

			//store resulting cost
			hipMemcpyAsync( (outputData[i])+voxel, device_WorkingBuffer, sizeof(float), hipMemcpyDeviceToHost, *stream );

		}
	}
	hipStreamSynchronize(*stream);

	//switch to entropy
	if( !useEntropy )
		for( int voxel = 0; voxel < VolumeSize; voxel++ )
			for( int i = 0; i < information.NumberOfLabels; i++)
				(outputData[i])[voxel] =  exp(-((outputData[i])[voxel]) );

	//remove allocated memory
	hipFree(device_KohonenMap);
	hipFree(device_InputData);
	hipFree(device_BaseBuffer);
	hipFree(device_WorkingBuffer);
	hipFree(device_ProbabilityBuffer);
}