#include "hip/hip_runtime.h"
#include "CUDA_voxelclassifier.h"
#include "vtkCudaCommon.h"
#include <float.h>
#include <iostream>

__constant__ Voxel_Classifier_Information info;
texture<short, 2, hipReadModeElementType> ClassifyPrimaryTexture;
texture<short, 2, hipReadModeElementType> ClassifyKeyholeTexture;

hipChannelFormatDesc Voxel_Classifier_ChannelDesc = hipCreateChannelDesc<short>();

__device__ bool WithinPlanes(const float* ConstantPlanes, const int NumPlanes, const int3& index)
{
  bool flag = false;
#pragma unroll 1
  for ( int i = 0; i < NumPlanes; i++ )
  {

    //collect all the information about the current clipping plane
    float4 clippingPlane;
    __syncthreads();
    clippingPlane.x  = ConstantPlanes[4*i];
    clippingPlane.y  = ConstantPlanes[4*i+1];
    clippingPlane.z  = ConstantPlanes[4*i+2];
    clippingPlane.w  = ConstantPlanes[4*i+3];
    __syncthreads();

    const float t = -(clippingPlane.x*index.x +
                      clippingPlane.y*index.y +
                      clippingPlane.z*index.z +
                      clippingPlane.w);

    //if the ray intersects the plane, set the start or end point to the intersection point
    flag |= (t > 0.0f);

  }//for

  return !flag;
}

__global__ void ClassifyVolume( const float2* inputVolume, short* outputVolume )
{
  //get the index of the thread in the volume
  int inIndex = CUDASTDOFFSET;
  int3 index;
  index.x = inIndex % info.VolumeSize[0];
  index.z = inIndex / info.VolumeSize[0];
  index.y = index.z % info.VolumeSize[1];
  index.z = index.z / info.VolumeSize[1];

  //get the values from the volume
  float2 value = inputVolume[inIndex];
  value.x = (float) info.TextureSize * (value.x - info.Intensity1Low) * info.Intensity1Multiplier;
  value.y = (float) info.TextureSize * (value.y - info.Intensity2Low) * info.Intensity2Multiplier;
  __syncthreads();

  //check if we are in the clipping and keyhole planes
  bool inClipping = (info.NumberOfClippingPlanes == 0 || WithinPlanes(info.ClippingPlanes, info.NumberOfClippingPlanes, index));
  bool inKeyhole = (info.NumberOfKeyholePlanes > 0 && WithinPlanes(info.KeyholePlanes, info.NumberOfKeyholePlanes, index));
  __syncthreads();

  //find the primary classification
  short classification = inClipping ? tex2D(ClassifyPrimaryTexture, value.x, value.y) : 0;
  classification = (inClipping && inKeyhole) ? - tex2D(ClassifyKeyholeTexture, value.x, value.y) : classification;

  //output the final classification
  if( inIndex < info.VolumeSize[0]*info.VolumeSize[1]*info.VolumeSize[2] )
  {
    outputVolume[inIndex] = classification;
  }
}

void CUDAalgo_classifyVoxels( float* inputData, short* inputPrimaryTexture, short* inputKeyholeTexture, int textureSize,
                              short* outputData, Voxel_Classifier_Information& information,
                              hipStream_t* stream )
{
  //copy information to GPU
  hipMemcpyToSymbolAsync(HIP_SYMBOL(info), &information, sizeof(Voxel_Classifier_Information) );
  int VolumeSize = information.VolumeSize[0]*information.VolumeSize[1]*information.VolumeSize[2];

  //translate input onto device
  float* dev_InputData = 0;
  hipMalloc( (void**) &dev_InputData, 2*sizeof(float)*VolumeSize );
  hipMemcpyAsync(dev_InputData,inputData, 2*sizeof(float)*VolumeSize,
                  hipMemcpyHostToDevice, *stream);

  //translate classification textures onto the device
  hipArray* PrimaryTextureArray = 0;
  hipMallocArray( &PrimaryTextureArray, &Voxel_Classifier_ChannelDesc, textureSize, textureSize);
  cudaMemcpyToArrayAsync(PrimaryTextureArray, 0, 0, inputPrimaryTexture,
                         sizeof(short)*textureSize*textureSize, hipMemcpyHostToDevice, *stream);
  hipArray* KeyholeTextureArray = 0;
  hipMallocArray( &KeyholeTextureArray, &Voxel_Classifier_ChannelDesc, textureSize, textureSize);
  cudaMemcpyToArrayAsync(KeyholeTextureArray, 0, 0, inputKeyholeTexture,
                         sizeof(short)*textureSize*textureSize, hipMemcpyHostToDevice, *stream);
  hipDeviceSynchronize();
  ClassifyPrimaryTexture.normalized = false;
  ClassifyPrimaryTexture.filterMode = hipFilterModePoint;
  ClassifyPrimaryTexture.addressMode[0] = hipAddressModeClamp;
  ClassifyPrimaryTexture.addressMode[1] = hipAddressModeClamp;
  hipBindTextureToArray(ClassifyPrimaryTexture, PrimaryTextureArray, Voxel_Classifier_ChannelDesc);
  ClassifyKeyholeTexture.normalized = false;
  ClassifyKeyholeTexture.filterMode = hipFilterModePoint;
  ClassifyKeyholeTexture.addressMode[0] = hipAddressModeClamp;
  ClassifyKeyholeTexture.addressMode[1] = hipAddressModeClamp;
  hipBindTextureToArray(ClassifyKeyholeTexture, KeyholeTextureArray, Voxel_Classifier_ChannelDesc);

  hipDeviceSynchronize();
  std::cout << "Load textures: " << std::endl << hipGetErrorString( hipGetLastError() ) << std::endl;

  //allocate working memory for the output
  short* dev_OutputData = 0;
  hipMalloc( (void**) &dev_OutputData, sizeof(short)*VolumeSize );

  //classify the volume
  dim3 grid = GetGrid( VolumeSize );
  dim3 threads(NUMTHREADS,1,1);
  ClassifyVolume<<< grid, threads, 0, *stream >>>((float2*)dev_InputData, dev_OutputData);

  hipDeviceSynchronize();
  std::cout << "Classify: " << hipGetErrorString( hipGetLastError() ) << std::endl;

  //retrieve classified output
  hipMemcpyAsync( outputData, dev_OutputData, sizeof(short)*VolumeSize,
                   hipMemcpyDeviceToHost, *stream);
  hipStreamSynchronize(*stream);

  hipDeviceSynchronize();
  std::cout << "Memcpy: " << hipGetErrorString( hipGetLastError() ) << std::endl;

  //deallocate textures and image memory
  hipFree( dev_InputData );
  hipUnbindTexture( ClassifyPrimaryTexture );
  hipUnbindTexture( ClassifyKeyholeTexture );
  hipFreeArray( PrimaryTextureArray );
  hipFreeArray( KeyholeTextureArray );
  hipFree( dev_OutputData );
}
