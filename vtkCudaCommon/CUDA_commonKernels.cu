#include "hip/hip_runtime.h"
#include "CUDA_commonKernels.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <float.h>

//---------------------------------------------------------------------------//
//------------------------COMMON CONFIG STATEMENTS---------------------------//
//---------------------------------------------------------------------------//

#define MAX_GRID_SIZE 65535

dim3 GetGrid(int size){
	dim3 grid( size, 1, 1 );
	if( grid.x > MAX_GRID_SIZE ) grid.x = grid.y = (int) sqrt( (double)(size-1) ) + 1;
	else if( grid.y > MAX_GRID_SIZE ) grid.x = grid.y = grid.z = (int) pow( (double)(size-1), (double)1.0/3.0 ) + 1;
	return grid;
}

//---------------------------------------------------------------------------//
//-------------------------COMMON UNARY OPERATORS----------------------------//
//---------------------------------------------------------------------------//

template<class T> 
__global__ void ZeroOutBuffer(T* buffer, int size){
	int offset = CUDASTDOFFSET;
	if(offset < size ) buffer[offset] = 0;
}
template __global__ void ZeroOutBuffer<char> (char* buffer, int size);
template __global__ void ZeroOutBuffer<signed char> (signed char* buffer, int size);
template __global__ void ZeroOutBuffer<unsigned char> (unsigned char* buffer, int size);
template __global__ void ZeroOutBuffer<short> (short* buffer, int size);
template __global__ void ZeroOutBuffer<unsigned short> (unsigned short* buffer, int size);
template __global__ void ZeroOutBuffer<int> (int* buffer, int size);
template __global__ void ZeroOutBuffer<unsigned int> (unsigned int* buffer, int size);
template __global__ void ZeroOutBuffer<long> (long* buffer, int size);
template __global__ void ZeroOutBuffer<unsigned long> (unsigned long* buffer, int size);
template __global__ void ZeroOutBuffer<float> (float* buffer, int size);
template __global__ void ZeroOutBuffer<double> (double* buffer, int size);
template __global__ void ZeroOutBuffer<long long> (long long* buffer, int size);
template __global__ void ZeroOutBuffer<unsigned long long> (unsigned long long* buffer, int size);

template<class T> 
__global__ void OneOutBuffer(T* buffer, int size){
	int offset = CUDASTDOFFSET;
	if(offset < size ) buffer[offset] = 1;
}
template __global__ void OneOutBuffer<char> (char* buffer, int size);
template __global__ void OneOutBuffer<signed char> (signed char* buffer, int size);
template __global__ void OneOutBuffer<unsigned char> (unsigned char* buffer, int size);
template __global__ void OneOutBuffer<short> (short* buffer, int size);
template __global__ void OneOutBuffer<unsigned short> (unsigned short* buffer, int size);
template __global__ void OneOutBuffer<int> (int* buffer, int size);
template __global__ void OneOutBuffer<unsigned int> (unsigned int* buffer, int size);
template __global__ void OneOutBuffer<long> (long* buffer, int size);
template __global__ void OneOutBuffer<unsigned long> (unsigned long* buffer, int size);
template __global__ void OneOutBuffer<float> (float* buffer, int size);
template __global__ void OneOutBuffer<double> (double* buffer, int size);
template __global__ void OneOutBuffer<long long> (long long* buffer, int size);
template __global__ void OneOutBuffer<unsigned long long> (unsigned long long* buffer, int size);

template<class T> 
__global__ void SetBufferToConst(T* buffer, T value, int size){
	int offset = CUDASTDOFFSET;
	if( offset < size ) buffer[offset] = value;
}
template __global__ void SetBufferToConst<char> (char* buffer, char value, int size);
template __global__ void SetBufferToConst<signed char> (signed char* buffer, signed char value, int size);
template __global__ void SetBufferToConst<unsigned char> (unsigned char* buffer, unsigned char value, int size);
template __global__ void SetBufferToConst<short> (short* buffer, short value, int size);
template __global__ void SetBufferToConst<unsigned short> (unsigned short* buffer, unsigned short value, int size);
template __global__ void SetBufferToConst<int> (int* buffer, int value, int size);
template __global__ void SetBufferToConst<unsigned int> (unsigned int* buffer, unsigned int value, int size);
template __global__ void SetBufferToConst<long> (long* buffer, long value, int size);
template __global__ void SetBufferToConst<unsigned long> (unsigned long* buffer, unsigned long value, int size);
template __global__ void SetBufferToConst<float> (float* buffer, float value, int size);
template __global__ void SetBufferToConst<double> (double* buffer, double value, int size);
template __global__ void SetBufferToConst<long long> (long long* buffer, long long value, int size);
template __global__ void SetBufferToConst<unsigned long long> (unsigned long long* buffer, unsigned long long value, int size);


template<class T>
__global__ void TranslateBuffer(T* buffer, T scale, T shift, int size){
	int offset = CUDASTDOFFSET;
	T value = scale * buffer[offset] + shift;
	if(offset < size ) buffer[offset] = value;
}
template __global__ void TranslateBuffer<char>(char* buffer, char scale, char shift, int size);
template __global__ void TranslateBuffer<signed char>(signed char* buffer, signed char scale, signed char shift, int size);
template __global__ void TranslateBuffer<unsigned char>(unsigned char* buffer, unsigned char scale, unsigned char shift, int size);
template __global__ void TranslateBuffer<short>(short* buffer, short scale, short shift, int size);
template __global__ void TranslateBuffer<unsigned short>(unsigned short* buffer, unsigned short scale, unsigned short shift, int size);
template __global__ void TranslateBuffer<int>(int* buffer, int scale, int shift, int size);
template __global__ void TranslateBuffer<unsigned int>(unsigned int* buffer, unsigned int scale, unsigned int shift, int size);
template __global__ void TranslateBuffer<long>(long* buffer, long scale, long shift, int size);
template __global__ void TranslateBuffer<unsigned long>(unsigned long* buffer, unsigned long scale, unsigned long shift, int size);
template __global__ void TranslateBuffer<float>(float* buffer, float scale, float shift, int size);
template __global__ void TranslateBuffer<double>(double* buffer, double scale, double shift, int size);
template __global__ void TranslateBuffer<long long>(long long* buffer, long long scale, long long shift, int size);
template __global__ void TranslateBuffer<unsigned long long>(unsigned long long* buffer, unsigned long long scale, unsigned long long shift, int size);

__global__ void ReplaceNANs(float* buffer, float value, int size){
	int offset = CUDASTDOFFSET;
	float current = buffer[offset];
	current = isfinite(current) ? current : value;
	if(offset < size ) buffer[offset] = current;
}

template<class T> 
__global__ void LogBuffer(T* buffer, int size){
	int offset = CUDASTDOFFSET;
	float input = (float) buffer[offset];
	T value = (T) log( input );
	if(offset < size ) buffer[offset] = value;
}
template __global__ void LogBuffer<char> (char* buffer, int size);
template __global__ void LogBuffer<signed char> (signed char* buffer, int size);
template __global__ void LogBuffer<unsigned char> (unsigned char* buffer, int size);
template __global__ void LogBuffer<short> (short* buffer, int size);
template __global__ void LogBuffer<unsigned short> (unsigned short* buffer, int size);
template __global__ void LogBuffer<int> (int* buffer, int size);
template __global__ void LogBuffer<unsigned int> (unsigned int* buffer, int size);
template __global__ void LogBuffer<long> (long* buffer, int size);
template __global__ void LogBuffer<unsigned long> (unsigned long* buffer, int size);
template __global__ void LogBuffer<float> (float* buffer, int size);
template __global__ void LogBuffer<double> (double* buffer, int size);
template __global__ void LogBuffer<long long> (long long* buffer, int size);
template __global__ void LogBuffer<unsigned long long> (unsigned long long* buffer, int size);

template<class T> 
__global__ void NegLogBuffer(T* buffer, int size){
	int offset = CUDASTDOFFSET;
	float input = (float) buffer[offset];
	T value = (T) -log( input );
	if(offset < size ) buffer[offset] = value;
}
template __global__ void NegLogBuffer<char> (char* buffer, int size);
template __global__ void NegLogBuffer<signed char> (signed char* buffer, int size);
template __global__ void NegLogBuffer<unsigned char> (unsigned char* buffer, int size);
template __global__ void NegLogBuffer<short> (short* buffer, int size);
template __global__ void NegLogBuffer<unsigned short> (unsigned short* buffer, int size);
template __global__ void NegLogBuffer<int> (int* buffer, int size);
template __global__ void NegLogBuffer<unsigned int> (unsigned int* buffer, int size);
template __global__ void NegLogBuffer<long> (long* buffer, int size);
template __global__ void NegLogBuffer<unsigned long> (unsigned long* buffer, int size);
template __global__ void NegLogBuffer<float> (float* buffer, int size);
template __global__ void NegLogBuffer<double> (double* buffer, int size);
template __global__ void NegLogBuffer<long long> (long long* buffer, int size);
template __global__ void NegLogBuffer<unsigned long long> (unsigned long long* buffer, int size);

template<class T, class S>
__global__ void IncrementBuffer(T* labelBuffer, T desiredLabel, S* agreement, int size){
	int idx = CUDASTDOFFSET;
	S newAgreement = agreement[idx];
	T labelValue = labelBuffer[idx];
	newAgreement += (labelValue == desiredLabel) ? 1 : 0;
	if( idx < size ) agreement[idx] = newAgreement;
}
template __global__ void IncrementBuffer<char, char> (char* labelBuffer, char desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<signed char, char> (signed char* labelBuffer, signed char desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, char> (unsigned char* labelBuffer, unsigned char desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<short, char> (short* labelBuffer, short desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, char> (unsigned short* labelBuffer, unsigned short desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<int, char> (int* labelBuffer, int desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, char> (unsigned int* labelBuffer, unsigned int desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<long, char> (long* labelBuffer, long desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, char> (unsigned long* labelBuffer, unsigned long desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<float, char> (float* labelBuffer, float desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<double, char> (double* labelBuffer, double desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<long long, char> (long long* labelBuffer, long long desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, char> (unsigned long long* labelBuffer, unsigned long long desiredLabel, char* agreement, int size);
template __global__ void IncrementBuffer<char, signed char> (char* labelBuffer, char desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<signed char, signed char> (signed char* labelBuffer, signed char desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, signed char> (unsigned char* labelBuffer, unsigned char desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<short, signed char> (short* labelBuffer, short desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, signed char> (unsigned short* labelBuffer, unsigned short desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<int, signed char> (int* labelBuffer, int desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, signed char> (unsigned int* labelBuffer, unsigned int desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<long, signed char> (long* labelBuffer, long desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, signed char> (unsigned long* labelBuffer, unsigned long desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<float, signed char> (float* labelBuffer, float desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<double, signed char> (double* labelBuffer, double desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<long long, signed char> (long long* labelBuffer, long long desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, signed char> (unsigned long long* labelBuffer, unsigned long long desiredLabel, signed char* agreement, int size);
template __global__ void IncrementBuffer<char, unsigned char> (char* labelBuffer, char desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<signed char, unsigned char> (signed char* labelBuffer, signed char desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, unsigned char> (unsigned char* labelBuffer, unsigned char desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<short, unsigned char> (short* labelBuffer, short desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, unsigned char> (unsigned short* labelBuffer, unsigned short desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<int, unsigned char> (int* labelBuffer, int desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, unsigned char> (unsigned int* labelBuffer, unsigned int desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<long, unsigned char> (long* labelBuffer, long desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, unsigned char> (unsigned long* labelBuffer, unsigned long desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<float, unsigned char> (float* labelBuffer, float desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<double, unsigned char> (double* labelBuffer, double desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<long long, unsigned char> (long long* labelBuffer, long long desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, unsigned char> (unsigned long long* labelBuffer, unsigned long long desiredLabel, unsigned char* agreement, int size);
template __global__ void IncrementBuffer<char, short> (char* labelBuffer, char desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<signed char, short> (signed char* labelBuffer, signed char desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, short> (unsigned char* labelBuffer, unsigned char desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<short, short> (short* labelBuffer, short desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, short> (unsigned short* labelBuffer, unsigned short desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<int, short> (int* labelBuffer, int desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, short> (unsigned int* labelBuffer, unsigned int desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<long, short> (long* labelBuffer, long desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, short> (unsigned long* labelBuffer, unsigned long desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<float, short> (float* labelBuffer, float desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<double, short> (double* labelBuffer, double desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<long long, short> (long long* labelBuffer, long long desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, short> (unsigned long long* labelBuffer, unsigned long long desiredLabel, short* agreement, int size);
template __global__ void IncrementBuffer<char, unsigned short> (char* labelBuffer, char desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<signed char, unsigned short> (signed char* labelBuffer, signed char desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, unsigned short> (unsigned char* labelBuffer, unsigned char desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<short, unsigned short> (short* labelBuffer, short desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, unsigned short> (unsigned short* labelBuffer, unsigned short desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<int, unsigned short> (int* labelBuffer, int desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, unsigned short> (unsigned int* labelBuffer, unsigned int desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<long, unsigned short> (long* labelBuffer, long desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, unsigned short> (unsigned long* labelBuffer, unsigned long desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<float, unsigned short> (float* labelBuffer, float desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<double, unsigned short> (double* labelBuffer, double desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<long long, unsigned short> (long long* labelBuffer, long long desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, unsigned short> (unsigned long long* labelBuffer, unsigned long long desiredLabel, unsigned short* agreement, int size);
template __global__ void IncrementBuffer<char, int> (char* labelBuffer, char desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<signed char, int> (signed char* labelBuffer, signed char desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, int> (unsigned char* labelBuffer, unsigned char desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<short, int> (short* labelBuffer, short desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, int> (unsigned short* labelBuffer, unsigned short desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<int, int> (int* labelBuffer, int desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, int> (unsigned int* labelBuffer, unsigned int desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<long, int> (long* labelBuffer, long desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, int> (unsigned long* labelBuffer, unsigned long desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<float, int> (float* labelBuffer, float desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<double, int> (double* labelBuffer, double desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<long long, int> (long long* labelBuffer, long long desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, int> (unsigned long long* labelBuffer, unsigned long long desiredLabel, int* agreement, int size);
template __global__ void IncrementBuffer<char, unsigned int> (char* labelBuffer, char desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<signed char, unsigned int> (signed char* labelBuffer, signed char desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, unsigned int> (unsigned char* labelBuffer, unsigned char desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<short, unsigned int> (short* labelBuffer, short desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, unsigned int> (unsigned short* labelBuffer, unsigned short desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<int, unsigned int> (int* labelBuffer, int desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, unsigned int> (unsigned int* labelBuffer, unsigned int desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<long, unsigned int> (long* labelBuffer, long desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, unsigned int> (unsigned long* labelBuffer, unsigned long desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<float, unsigned int> (float* labelBuffer, float desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<double, unsigned int> (double* labelBuffer, double desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<long long, unsigned int> (long long* labelBuffer, long long desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, unsigned int> (unsigned long long* labelBuffer, unsigned long long desiredLabel, unsigned int* agreement, int size);
template __global__ void IncrementBuffer<char, long> (char* labelBuffer, char desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<signed char, long> (signed char* labelBuffer, signed char desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, long> (unsigned char* labelBuffer, unsigned char desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<short, long> (short* labelBuffer, short desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, long> (unsigned short* labelBuffer, unsigned short desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<int, long> (int* labelBuffer, int desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, long> (unsigned int* labelBuffer, unsigned int desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<long, long> (long* labelBuffer, long desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, long> (unsigned long* labelBuffer, unsigned long desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<float, long> (float* labelBuffer, float desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<double, long> (double* labelBuffer, double desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<long long, long> (long long* labelBuffer, long long desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, long> (unsigned long long* labelBuffer, unsigned long long desiredLabel, long* agreement, int size);
template __global__ void IncrementBuffer<char, unsigned long> (char* labelBuffer, char desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<signed char, unsigned long> (signed char* labelBuffer, signed char desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, unsigned long> (unsigned char* labelBuffer, unsigned char desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<short, unsigned long> (short* labelBuffer, short desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, unsigned long> (unsigned short* labelBuffer, unsigned short desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<int, unsigned long> (int* labelBuffer, int desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, unsigned long> (unsigned int* labelBuffer, unsigned int desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<long, unsigned long> (long* labelBuffer, long desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, unsigned long> (unsigned long* labelBuffer, unsigned long desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<float, unsigned long> (float* labelBuffer, float desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<double, unsigned long> (double* labelBuffer, double desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<long long, unsigned long> (long long* labelBuffer, long long desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, unsigned long> (unsigned long long* labelBuffer, unsigned long long desiredLabel, unsigned long* agreement, int size);
template __global__ void IncrementBuffer<char, float> (char* labelBuffer, char desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<signed char, float> (signed char* labelBuffer, signed char desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, float> (unsigned char* labelBuffer, unsigned char desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<short, float> (short* labelBuffer, short desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, float> (unsigned short* labelBuffer, unsigned short desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<int, float> (int* labelBuffer, int desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, float> (unsigned int* labelBuffer, unsigned int desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<long, float> (long* labelBuffer, long desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, float> (unsigned long* labelBuffer, unsigned long desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<float, float> (float* labelBuffer, float desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<double, float> (double* labelBuffer, double desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<long long, float> (long long* labelBuffer, long long desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, float> (unsigned long long* labelBuffer, unsigned long long desiredLabel, float* agreement, int size);
template __global__ void IncrementBuffer<char, double> (char* labelBuffer, char desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<signed char, double> (signed char* labelBuffer, signed char desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, double> (unsigned char* labelBuffer, unsigned char desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<short, double> (short* labelBuffer, short desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, double> (unsigned short* labelBuffer, unsigned short desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<int, double> (int* labelBuffer, int desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, double> (unsigned int* labelBuffer, unsigned int desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<long, double> (long* labelBuffer, long desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, double> (unsigned long* labelBuffer, unsigned long desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<float, double> (float* labelBuffer, float desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<double, double> (double* labelBuffer, double desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<long long, double> (long long* labelBuffer, long long desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, double> (unsigned long long* labelBuffer, unsigned long long desiredLabel, double* agreement, int size);
template __global__ void IncrementBuffer<char, long long> (char* labelBuffer, char desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<signed char, long long> (signed char* labelBuffer, signed char desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, long long> (unsigned char* labelBuffer, unsigned char desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<short, long long> (short* labelBuffer, short desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, long long> (unsigned short* labelBuffer, unsigned short desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<int, long long> (int* labelBuffer, int desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, long long> (unsigned int* labelBuffer, unsigned int desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<long, long long> (long* labelBuffer, long desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, long long> (unsigned long* labelBuffer, unsigned long desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<float, long long> (float* labelBuffer, float desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<double, long long> (double* labelBuffer, double desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<long long, long long> (long long* labelBuffer, long long desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, long long> (unsigned long long* labelBuffer, unsigned long long desiredLabel, long long* agreement, int size);
template __global__ void IncrementBuffer<char, unsigned long long> (char* labelBuffer, char desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<signed char, unsigned long long> (signed char* labelBuffer, signed char desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<unsigned char, unsigned long long> (unsigned char* labelBuffer, unsigned char desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<short, unsigned long long> (short* labelBuffer, short desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<unsigned short, unsigned long long> (unsigned short* labelBuffer, unsigned short desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<int, unsigned long long> (int* labelBuffer, int desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<unsigned int, unsigned long long> (unsigned int* labelBuffer, unsigned int desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<long, unsigned long long> (long* labelBuffer, long desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<unsigned long, unsigned long long> (unsigned long* labelBuffer, unsigned long desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<float, unsigned long long> (float* labelBuffer, float desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<double, unsigned long long> (double* labelBuffer, double desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<long long, unsigned long long> (long long* labelBuffer, long long desiredLabel, unsigned long long* agreement, int size);
template __global__ void IncrementBuffer<unsigned long long, unsigned long long> (unsigned long long* labelBuffer, unsigned long long desiredLabel, unsigned long long* agreement, int size);

__global__ void SetBufferToRandom(float* buffer, float min, float max, int size){
	int offset = CUDASTDOFFSET;
	hiprandState localState;
	hiprand_init(7+offset, offset, 0, &localState);
	__syncthreads();

	float value = min + (max-min)*hiprand_uniform(&localState);
	if(offset < size ) buffer[offset] = value;
}

//---------------------------------------------------------------------------//
//-------------------------COMMON BINARY OPERATORS---------------------------//
//---------------------------------------------------------------------------//

template<class T> 
__global__ void SumBuffers(T* outBuffer, T* sumBuffer, int size){
	int offset = CUDASTDOFFSET;
	T value = outBuffer[offset] + sumBuffer[offset];
	if(offset < size ) outBuffer[offset] = value;
}
template __global__ void SumBuffers<char>(char* buffer1, char* buffer2, int size);
template __global__ void SumBuffers<signed char>(signed char* buffer1, signed char* buffer2, int size);
template __global__ void SumBuffers<unsigned char>(unsigned char* buffer1, unsigned char* buffer2, int size);
template __global__ void SumBuffers<short>(short* buffer1, short* buffer2, int size);
template __global__ void SumBuffers<unsigned short>(unsigned short* buffer1, unsigned short* buffer2, int size);
template __global__ void SumBuffers<int>(int* buffer1, int* buffer2, int size);
template __global__ void SumBuffers<unsigned int>(unsigned int* buffer1, unsigned int* buffer2, int size);
template __global__ void SumBuffers<long>(long* buffer1, long* buffer2, int size);
template __global__ void SumBuffers<unsigned long>(unsigned long* buffer1, unsigned long* buffer2, int size);
template __global__ void SumBuffers<float>(float* buffer1, float* buffer2, int size);
template __global__ void SumBuffers<double>(double* buffer1, double* buffer2, int size);
template __global__ void SumBuffers<long long>(long long* buffer1, long long* buffer2, int size);
template __global__ void SumBuffers<unsigned long long>(unsigned long long* buffer1, unsigned long long* buffer2, int size);

template<class T> 
__global__ void CopyBuffers(T* outBuffer, T* inBuffer, int size){
	int offset = CUDASTDOFFSET;
	T value = inBuffer[offset];
	if(offset < size ) outBuffer[offset] = value;
}
template __global__ void CopyBuffers<char>(char* buffer1, char* buffer2, int size);
template __global__ void CopyBuffers<signed char>(signed char* buffer1, signed char* buffer2, int size);
template __global__ void CopyBuffers<unsigned char>(unsigned char* buffer1, unsigned char* buffer2, int size);
template __global__ void CopyBuffers<short>(short* buffer1, short* buffer2, int size);
template __global__ void CopyBuffers<unsigned short>(unsigned short* buffer1, unsigned short* buffer2, int size);
template __global__ void CopyBuffers<int>(int* buffer1, int* buffer2, int size);
template __global__ void CopyBuffers<unsigned int>(unsigned int* buffer1, unsigned int* buffer2, int size);
template __global__ void CopyBuffers<long>(long* buffer1, long* buffer2, int size);
template __global__ void CopyBuffers<unsigned long>(unsigned long* buffer1, unsigned long* buffer2, int size);
template __global__ void CopyBuffers<float>(float* buffer1, float* buffer2, int size);
template __global__ void CopyBuffers<double>(double* buffer1, double* buffer2, int size);
template __global__ void CopyBuffers<long long>(long long* buffer1, long long* buffer2, int size);
template __global__ void CopyBuffers<unsigned long long>(unsigned long long* buffer1, unsigned long long* buffer2, int size);

template<class T>
__global__ void MultiplyBuffers(T* outBuffer, T* multBuffer, T scale, T shift, int size){
	int offset = CUDASTDOFFSET;
	float value = (scale * outBuffer[offset] + shift) * multBuffer[offset];
	if(offset < size ) outBuffer[offset] = value;
}
template __global__ void MultiplyBuffers<char>(char* buffer1, char* buffer2, char value1, char value2, int size);
template __global__ void MultiplyBuffers<signed char>(signed char* buffer1, signed char* buffer2, signed char value1, signed char value2, int size);
template __global__ void MultiplyBuffers<unsigned char>(unsigned char* buffer1, unsigned char* buffer2, unsigned char value1, unsigned char value2, int size);
template __global__ void MultiplyBuffers<short>(short* buffer1, short* buffer2, short value1, short value2, int size);
template __global__ void MultiplyBuffers<unsigned short>(unsigned short* buffer1, unsigned short* buffer2, unsigned short value1, unsigned short value2, int size);
template __global__ void MultiplyBuffers<int>(int* buffer1, int* buffer2, int value1, int value2, int size);
template __global__ void MultiplyBuffers<unsigned int>(unsigned int* buffer1, unsigned int* buffer2, unsigned int value1, unsigned int value2, int size);
template __global__ void MultiplyBuffers<long>(long* buffer1, long* buffer2, long value1, long value2, int size);
template __global__ void MultiplyBuffers<unsigned long>(unsigned long* buffer1, unsigned long* buffer2, unsigned long value1, unsigned long value2, int size);
template __global__ void MultiplyBuffers<float>(float* buffer1, float* buffer2, float value1, float value2, int size);
template __global__ void MultiplyBuffers<double>(double* buffer1, double* buffer2, double value1, double value2, int size);
template __global__ void MultiplyBuffers<long long>(long long* buffer1, long long* buffer2, long long value1, long long value2, int size);
template __global__ void MultiplyBuffers<unsigned long long>(unsigned long long* buffer1, unsigned long long* buffer2, unsigned long long value1, unsigned long long value2, int size);

template<class T>
__global__ void MultiplyBuffers(T* outBuffer, T* multBuffer, int size){
	int offset = CUDASTDOFFSET;
	float value = outBuffer[offset] * multBuffer[offset];
	if(offset < size ) outBuffer[offset] = value;
}
template __global__ void MultiplyBuffers<char>(char* buffer1, char* buffer2, int size);
template __global__ void MultiplyBuffers<signed char>(signed char* buffer1, signed char* buffer2, int size);
template __global__ void MultiplyBuffers<unsigned char>(unsigned char* buffer1, unsigned char* buffer2, int size);
template __global__ void MultiplyBuffers<short>(short* buffer1, short* buffer2, int size);
template __global__ void MultiplyBuffers<unsigned short>(unsigned short* buffer1, unsigned short* buffer2, int size);
template __global__ void MultiplyBuffers<int>(int* buffer1, int* buffer2, int size);
template __global__ void MultiplyBuffers<unsigned int>(unsigned int* buffer1, unsigned int* buffer2, int size);
template __global__ void MultiplyBuffers<long>(long* buffer1, long* buffer2, int size);
template __global__ void MultiplyBuffers<unsigned long>(unsigned long* buffer1, unsigned long* buffer2, int size);
template __global__ void MultiplyBuffers<float>(float* buffer1, float* buffer2, int size);
template __global__ void MultiplyBuffers<double>(double* buffer1, double* buffer2, int size);
template __global__ void MultiplyBuffers<long long>(long long* buffer1, long long* buffer2, int size);
template __global__ void MultiplyBuffers<unsigned long long>(unsigned long long* buffer1, unsigned long long* buffer2, int size);

template<class T>
__global__ void MultiplyAndStoreBuffer(T* inBuffer, T* outBuffer, T number, int size){
	int idx = CUDASTDOFFSET;
	T value = inBuffer[idx] * number;
	if( idx < size ) outBuffer[idx] = value;
}
template __global__ void MultiplyAndStoreBuffer<char>(char* buffer1, char* buffer2, char value, int size);
template __global__ void MultiplyAndStoreBuffer<signed char>(signed char* buffer1, signed char* buffer2, signed char value, int size);
template __global__ void MultiplyAndStoreBuffer<unsigned char>(unsigned char* buffer1, unsigned char* buffer2, unsigned char value, int size);
template __global__ void MultiplyAndStoreBuffer<short>(short* buffer1, short* buffer2, short value, int size);
template __global__ void MultiplyAndStoreBuffer<unsigned short>(unsigned short* buffer1, unsigned short* buffer2, unsigned short value, int size);
template __global__ void MultiplyAndStoreBuffer<int>(int* buffer1, int* buffer2, int value, int size);
template __global__ void MultiplyAndStoreBuffer<unsigned int>(unsigned int* buffer1, unsigned int* buffer2, unsigned int value, int size);
template __global__ void MultiplyAndStoreBuffer<long>(long* buffer1, long* buffer2, long value, int size);
template __global__ void MultiplyAndStoreBuffer<unsigned long>(unsigned long* buffer1, unsigned long* buffer2, unsigned long value, int size);
template __global__ void MultiplyAndStoreBuffer<float>(float* buffer1, float* buffer2, float value, int size);
template __global__ void MultiplyAndStoreBuffer<double>(double* buffer1, double* buffer2, double value, int size);
template __global__ void MultiplyAndStoreBuffer<long long>(long long* buffer1, long long* buffer2, long long value, int size);
template __global__ void MultiplyAndStoreBuffer<unsigned long long>(unsigned long long* buffer1, unsigned long long* buffer2, unsigned long long value, int size);


//---------------------------------------------------------------------------//
//----------------------------COMMON ACCUMULATORS----------------------------//
//---------------------------------------------------------------------------//


void SumData(int size, int threads, int blocks, float* dataBuffer, hipStream_t* stream ){

    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    int smemSize = (threads <= 32) ? 2 * threads * (sizeof(float)+sizeof(short2)) : threads * (sizeof(float)+sizeof(short2));
	switch (threads)
	{
	case 512:
		SumOverSmallBuffer<512><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 256:
		SumOverSmallBuffer<256><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 128:
		SumOverSmallBuffer<128><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 64:
		SumOverSmallBuffer< 64><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 32:
		SumOverSmallBuffer< 32><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 16:
		SumOverSmallBuffer< 16><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 8:
		SumOverSmallBuffer< 8><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 4:
		SumOverSmallBuffer< 4><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 2:
		SumOverSmallBuffer< 2><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 1:
		SumOverSmallBuffer< 1><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	}

}

template <unsigned int blockSize>
__global__ void SumOverSmallBuffer(float *buffer, unsigned int n)
{
	__shared__ float sdata[blockSize];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;
	sdata[tid] = 0.0f;
	
	while (i < n) {
		sdata[tid] += buffer[i];
		sdata[tid] += buffer[i+blockSize];
		i += gridSize;
		__syncthreads();
	}
	
	if (blockSize >= 512) { if (tid < 256) {
			sdata[tid] += sdata[tid + 256];
	} __syncthreads(); }

	if (blockSize >= 256) { if (tid < 128) {
			sdata[tid] += sdata[tid + 128];
	} __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) {
			sdata[tid] += sdata[tid + 64];
	} __syncthreads(); }
	
	if (tid < 32) {
		if (blockSize >= 64){
			sdata[tid] += sdata[tid + 32];
			__syncthreads();
		}
		if (blockSize >= 32){
			sdata[tid] += sdata[tid + 16];
			__syncthreads();
		}
		if (blockSize >= 16){
			sdata[tid] += sdata[tid + 8];
			__syncthreads();
		}
		if (blockSize >=  8){
			sdata[tid] += sdata[tid + 4];
			__syncthreads();
		}
		if (blockSize >=  4){
			sdata[tid] += sdata[tid + 2];
			__syncthreads();
		}
		if (blockSize >=  2){
			sdata[tid] += sdata[tid + 1];
			__syncthreads();
		}
	}
	if (tid == 0){
		buffer[0] = sdata[0];
	}
}

__global__ void SumOverLargeBuffer( float* buffer, int spread, int size ){
	
	int offset = CUDASTDOFFSET;
	float value1 = buffer[offset];
	float value2 = buffer[offset+spread];

	if( offset+spread < size )
		buffer[offset] = value1+value2;

}

#define Logariture(value1, value2)	0.5f * ((isfinite(value1 + log(1.0f + exp(value2-value1)))?value1 + log(1.0f + exp(value2-value1)):value2 + log(1.0f + exp(value1-value2))) + \
											(isfinite(value2 + log(1.0f + exp(value1-value2)))?value2 + log(1.0f + exp(value1-value2)):value1 + log(1.0f + exp(value2-value1))) )

void LogaritureData(int size, int threads, int blocks, float* dataBuffer, hipStream_t* stream ){

    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    int smemSize = (threads <= 32) ? 2 * threads * (sizeof(float)+sizeof(short2)) : threads * (sizeof(float)+sizeof(short2));
	switch (threads)
	{
	case 512:
		LogaritureOverSmallBuffer<512><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 256:
		LogaritureOverSmallBuffer<256><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 128:
		LogaritureOverSmallBuffer<128><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 64:
		LogaritureOverSmallBuffer< 64><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 32:
		LogaritureOverSmallBuffer< 32><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 16:
		LogaritureOverSmallBuffer< 16><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 8:
		LogaritureOverSmallBuffer< 8><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 4:
		LogaritureOverSmallBuffer< 4><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 2:
		LogaritureOverSmallBuffer< 2><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	case 1:
		LogaritureOverSmallBuffer< 1><<< dimGrid, dimBlock, smemSize, *stream >>>(dataBuffer, size); break;
	}

}

template <unsigned int blockSize>
__global__ void LogaritureOverSmallBuffer(float *buffer, unsigned int n)
{
	__shared__ float sdata[blockSize];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;
	sdata[tid] = -2.0f * FLT_MAX;
	
	while (i < n) {
		sdata[tid] = Logariture(sdata[tid], buffer[i]);
		sdata[tid] = Logariture(sdata[tid], buffer[i+blockSize]);
		i += gridSize;
		__syncthreads();
	}
	
	if (blockSize >= 512) { if (tid < 256) {
			sdata[tid] = Logariture(sdata[tid], sdata[tid + 256]);
	} __syncthreads(); }

	if (blockSize >= 256) { if (tid < 128) {
			sdata[tid] = Logariture(sdata[tid], sdata[tid + 128]);
	} __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) {
			sdata[tid] = Logariture(sdata[tid], sdata[tid + 64]);
	} __syncthreads(); }
	
	if (tid < 32) {
		if (blockSize >= 64){
			sdata[tid] = Logariture(sdata[tid], sdata[tid + 32]);
			__syncthreads();
		}
		if (blockSize >= 32){
			sdata[tid] = Logariture(sdata[tid], sdata[tid + 16]);
			__syncthreads();
		}
		if (blockSize >= 16){
			sdata[tid] = Logariture(sdata[tid], sdata[tid + 8]);
			__syncthreads();
		}
		if (blockSize >=  8){
			sdata[tid] = Logariture(sdata[tid], sdata[tid + 4]);
			__syncthreads();
		}
		if (blockSize >=  4){
			sdata[tid] = Logariture(sdata[tid], sdata[tid + 2]);
			__syncthreads();
		}
		if (blockSize >=  2){
			sdata[tid] = Logariture(sdata[tid], sdata[tid + 1]);
			__syncthreads();
		}
	}
	if (tid == 0){
		buffer[0] = sdata[0];
	}
}

__global__ void LogaritureOverLargeBuffer( float* buffer, int spread, int size ){
	
	int offset = CUDASTDOFFSET;
	float value1 = buffer[offset];
	float value2 = buffer[offset+spread];
	
	float result = Logariture(value1, value2);

	if( offset+spread < size )
		buffer[offset] = result;

}